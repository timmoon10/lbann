#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// Copyright (c) 2014-2019, Lawrence Livermore National Security, LLC.
// Produced at the Lawrence Livermore National Laboratory.
// Written by the LBANN Research Team (B. Van Essen, et al.) listed in
// the CONTRIBUTORS file. <lbann-dev@llnl.gov>
//
// LLNL-CODE-697807.
// All rights reserved.
//
// This file is part of LBANN: Livermore Big Artificial Neural Network
// Toolkit. For details, see http://software.llnl.gov/LBANN or
// https://github.com/LLNL/LBANN.
//
// Licensed under the Apache License, Version 2.0 (the "Licensee"); you
// may not use this file except in compliance with the License.  You may
// obtain a copy of the License at:
//
// http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or
// implied. See the License for the specific language governing
// permissions and limitations under the license.
////////////////////////////////////////////////////////////////////////////////

#define LBANN_SLICE_LAYER_INSTANTIATE
#include "lbann/layers/transform/slice.hpp"
#include "lbann/utils/cuda.hpp"

namespace lbann {

namespace {

using dim4 = cuda::array<size_t, 4>;

/**
 *  Block dimensions: bsize x 1 x 1
 *
 *  Grid dimensions: (max_input_dims[3] / bsize) x max_input_dims[2] x max_input_dims[1]
 */
template <typename T>
__global__ void concat4d_kernel(
  size_t num_inputs,
  const T* __restrict__ * __restrict__ input_buffer_list,
  const dim4* __restrict__ input_dims_list,
  const dim4* __restrict__ input_strides_list,
  T* __restrict__ output_buffer,
  dim4 output_strides,
  const size_t* __restrict__ output_offset_list) {

  // Indices
  const size_t gidx = threadIdx.x + blockIdx.x * blockDim.x;
  const size_t gidy = threadIdx.y + blockIdx.y * blockDim.y;
  const size_t gidz = threadIdx.z + blockIdx.z * blockDim.z;
  const size_t nthreadsx = gridDim.x * blockDim.x;
  const size_t nthreadsy = gridDim.y * blockDim.y;
  const size_t nthreadsz = gridDim.z * blockDim.z;

  for (size_t j=0; j<num_inputs; ++j) {

    // Current input tensor
    const auto& input_buffer = input_buffer_list[j];
    const auto& input_dims = input_dims_list[j];
    const auto& input_strides = input_strides_list[j];
    const auto& output_offset = output_offset_list[j];

    // Copy from input tensor to output tensor
    for (size_t i0=0; i0<input_dims[0]; ++i0) {
      for (size_t i1=gidz; i1<input_dims[1]; i1+=nthreadsz) {
        for (size_t i2=gidy; i2<input_dims[2]; i2+=nthreadsy) {
          for (size_t i3=gidx; i3<input_dims[3]; i3+=nthreadsx) {
            const auto& x = input_buffer[i0 * input_strides[0]
                                         + i1 * input_strides[1]
                                         + i2 * input_strides[2]
                                         + i3 * input_strides[3]];
            auto& y = output_buffer[output_offset
                                    + i0 * output_strides[0]
                                    + i1 * output_strides[1]
                                    + i2 * output_strides[2]
                                    + i3 * output_strides[3]];
            y = x;
          }
        }
      }
    }

  }

}

/**
 *  Block dimensions: bsize x 1 x 1
 *
 *  Grid dimensions: (max_output_dims[3] / bsize) x max_output_dims[2] x max_output_dims[1]
 *
 */
template <typename T>
__global__ void slice4d_kernel(
  size_t num_outputs,
  const T* __restrict__ input_buffer,
  dim4 input_strides,
  const size_t* __restrict__ input_offset_list,
  T* __restrict__ * __restrict__ output_buffer_list,
  const dim4* __restrict__ output_dims_list,
  const dim4* __restrict__ output_strides_list) {

  // Indices
  const size_t gidx = threadIdx.x + blockIdx.x * blockDim.x;
  const size_t gidy = threadIdx.y + blockIdx.y * blockDim.y;
  const size_t gidz = threadIdx.z + blockIdx.z * blockDim.z;
  const size_t nthreadsx = gridDim.x * blockDim.x;
  const size_t nthreadsy = gridDim.y * blockDim.y;
  const size_t nthreadsz = gridDim.z * blockDim.z;

  for (size_t j=0; j<num_outputs; ++j) {

    // Current output tensor
    const auto& input_offset = input_offset_list[j];
    auto& output_buffer = output_buffer_list[j];
    const auto& output_dims = output_dims_list[j];
    const auto& output_strides = output_strides_list[j];

    // Copy from input tensor to output tensor
    for (size_t i0=0; i0<output_dims[0]; ++i0) {
      for (size_t i1=gidz; i1<output_dims[1]; i1+=nthreadsz) {
        for (size_t i2=gidy; i2<output_dims[2]; i2+=nthreadsy) {
          for (size_t i3=gidx; i3<output_dims[3]; i3+=nthreadsx) {
            const auto& x = input_buffer[input_offset
                                         + i0 * input_strides[0]
                                         + i1 * input_strides[1]
                                         + i2 * input_strides[2]
                                         + i3 * input_strides[3]];
            auto& y = output_buffer[i0 * output_strides[0]
                                    + i1 * output_strides[1]
                                    + i2 * output_strides[2]
                                    + i3 * output_strides[3]];
            y = x;
          }
        }
      }
    }

  }

}

} // namespace <anon>

template <typename TensorDataType>
void fp_compute_impl(
  slice_layer<TensorDataType,data_layout::MODEL_PARALLEL,El::Device::GPU>& l) {
  // Tensor views have already been setup in fp_setup_outputs
}

template <typename TensorDataType>
void bp_compute_impl(
  slice_layer<TensorDataType,data_layout::MODEL_PARALLEL,El::Device::GPU>& l) {

  // Stack Elemental matrices on top of each other
  // Note: Assume each mini-batch sample is flat.
  auto& input_grad = l.get_error_signals();
  std::unique_ptr<El::AbstractDistMatrix<TensorDataType>> input_grad_v(
    input_grad.Construct(input_grad.Grid(), input_grad.Root()));
  size_t offset = l.m_slice_points.front();
  for (size_t j=0; j<static_cast<size_t>(l.get_num_children()); ++j) {
    const auto& output_grad = l.get_prev_error_signals(j);
    El::View(*input_grad_v, input_grad,
             El::IR(offset, offset+output_grad.Height()), El::ALL);
    El::Copy(output_grad, *input_grad_v);
    offset += output_grad.Height();
  }

}

template <typename TensorDataType>
void fp_compute_impl(
  slice_layer<TensorDataType,data_layout::DATA_PARALLEL,El::Device::GPU>& l) {

  // Check that number of dimensions is valid
  /// @todo Support tensors with arbitrary number of dimensions
  const auto& input_dims = l.get_input_dims();
  const size_t num_dims = input_dims.size();
  if (num_dims > 3) {
    LBANN_ERROR(l.get_type()," layer \"",l.get_name(),"\" ",
                "is operating on ",num_dims,"-D tensors, ",
                "but only 3-D tensors are currently supported");
  }

  // Get synchronization info from input tensor
  using LocalMatrix = El::Matrix<TensorDataType, El::Device::GPU>;
  const auto& input = l.get_prev_activations();
  const auto& local_input = dynamic_cast<const LocalMatrix&>(input.LockedMatrix());
  auto&& sync_info = El::SyncInfoFromMatrix(local_input);
  auto&& stream = sync_info.stream_;

  // Get dimensions and strides for each output tensor
  const size_t num_outputs = l.get_num_children();
  std::vector<TensorDataType*> output_buffer_list;
  std::vector<dim4> output_dims_list, output_strides_list;
  dim4 max_output_dims{0,0,0,0};
  for (size_t j=0; j<num_outputs; ++j) {
    auto& output = l.get_activations(j);
    const auto& output_dims = l.get_output_dims(j);

    // Construct dimensions and strides in reverse order
    // Note: Assume each mini-batch sample is fully packed.
    std::vector<size_t> rdims(output_dims.rbegin(), output_dims.rend());
    std::vector<size_t> rstrides(output_dims.size(), 1);
    for (size_t d=1; d<output_dims.size(); ++d) {
      rstrides[d] = rdims[d-1] * rstrides[d-1];
    }
    rdims.push_back(output.LocalWidth());
    rstrides.push_back(output.LDim());

    // Pad tensor dimensions to 4D
    rdims.resize(4, 1);
    rstrides.resize(4, rstrides.back());

    output_buffer_list.push_back(output.Buffer());
    output_dims_list.push_back({rdims[3], rdims[2], rdims[1], rdims[0]});
    output_strides_list.push_back(
      {rstrides[3], rstrides[2], rstrides[1], rstrides[0]});
    for (size_t i=0; i<4; ++i) {
      max_output_dims[i] = std::max(max_output_dims[i], rdims[3-i]);
    }
  }

  // Get strides for input tensor
  dim4 input_strides;
  {

    // Construct dimensions and strides in reverse order
    // Note: Assume each mini-batch sample is fully packed.
    std::vector<size_t> rdims(input_dims.rbegin(), input_dims.rend());
    std::vector<size_t> rstrides(input_dims.size(), 1);
    for (size_t d=1; d<input_dims.size(); ++d) {
      rstrides[d] = rdims[d-1] * rstrides[d-1];
    }
    rdims.push_back(local_input.Width());
    rstrides.push_back(local_input.LDim());

    // Pad tensor dimensions to 4D
    rdims.resize(4, 1);
    rstrides.resize(4, rstrides.back());

    input_strides = {rstrides[3], rstrides[2], rstrides[1], rstrides[0]};
  }

  // Compute each output tensor's offset in input tensor
  const size_t slice_dim_stride = input_strides[l.m_slice_dim+(4-num_dims)];
  std::vector<size_t> input_offset_list;
  for (const auto& slice_point : l.m_slice_points) {
    input_offset_list.push_back(slice_point * slice_dim_stride);
  }

  // Pack tensor data into a CPU buffer
  l.m_workspace_event.synchronize();
  l.m_workspace.resize(
    sizeof(size_t) * input_offset_list.size()
    + sizeof(TensorDataType*) * output_buffer_list.size()
    + sizeof(dim4) * output_dims_list.size()
    + sizeof(dim4) * output_strides_list.size());
  size_t pos = 0;
  std::memcpy(&l.m_workspace[pos], input_offset_list.data(),
              sizeof(size_t) * input_offset_list.size());
  pos += sizeof(size_t) * input_offset_list.size();
  std::memcpy(&l.m_workspace[pos], output_buffer_list.data(),
              sizeof(TensorDataType*) * output_buffer_list.size());
  pos += sizeof(TensorDataType*) * output_buffer_list.size();
  std::memcpy(&l.m_workspace[pos], output_dims_list.data(),
              sizeof(dim4) * output_dims_list.size());
  pos += sizeof(dim4) * output_dims_list.size();
  std::memcpy(&l.m_workspace[pos], output_strides_list.data(),
              sizeof(dim4) * output_strides_list.size());
  pos += sizeof(dim4) * output_strides_list.size();

  // Copy tensor data to GPU
  hydrogen::simple_buffer<unsigned char, El::Device::GPU> device_workspace(
    l.m_workspace.size(),
    sync_info);
  unsigned char* device_workspace_ptr = device_workspace.data();
  hipMemcpyAsync(device_workspace_ptr,
                  l.m_workspace.data(),
                  l.m_workspace.size(),
                  hipMemcpyHostToDevice,
                  stream);
  l.m_workspace_event.record(stream);
  pos = 0;
  auto&& device_input_offset_list
    = reinterpret_cast<const size_t*>(device_workspace_ptr+pos);
  pos += sizeof(size_t) * input_offset_list.size();
  auto&& device_output_buffer_list
    = reinterpret_cast<TensorDataType**>(device_workspace_ptr+pos);
  pos += sizeof(TensorDataType*) * output_buffer_list.size();
  auto&& device_output_dims_list
    = reinterpret_cast<const dim4*>(device_workspace_ptr+pos);
  pos += sizeof(dim4) * output_dims_list.size();
  auto&& device_output_strides_list
    = reinterpret_cast<const dim4*>(device_workspace_ptr+pos);
  pos += sizeof(dim4) * output_strides_list.size();

  // Launch CUDA kernel
  const auto& max_output_size = (max_output_dims[0] * max_output_dims[1]
                                 * max_output_dims[2] * max_output_dims[3]);
  if (max_output_size > 0) {
    constexpr size_t block_size = 64;
    dim3 block_dims, grid_dims;
    block_dims.x = block_size;
    grid_dims.x = (max_output_dims[3] + block_size - 1) / block_size;
    grid_dims.y = max_output_dims[2];
    grid_dims.z = max_output_dims[1];
    slice4d_kernel<<<grid_dims, block_dims, 0, stream>>>(
      num_outputs,
      local_input.LockedBuffer(),
      input_strides,
      device_input_offset_list,
      device_output_buffer_list,
      device_output_dims_list,
      device_output_strides_list);
  }

}

template <typename TensorDataType>
void bp_compute_impl(
  slice_layer<TensorDataType,data_layout::DATA_PARALLEL,El::Device::GPU>& l) {

  // Check that number of dimensions is valid
  /// @todo Support tensors with arbitrary number of dimensions
  const auto& input_dims = l.get_input_dims();
  const size_t num_dims = input_dims.size();
  if (num_dims > 3) {
    LBANN_ERROR(l.get_type()," layer \"",l.get_name(),"\" ",
                "is operating on ",num_dims,"-D tensors, ",
                "but only 3-D tensors are currently supported");
  }

  // Get synchronization info from input gradient tensor
  using LocalMatrix = El::Matrix<TensorDataType, El::Device::GPU>;
  auto& input_grad = l.get_error_signals();
  auto& local_input_grad = dynamic_cast<LocalMatrix&>(input_grad.Matrix());
  auto&& sync_info = El::SyncInfoFromMatrix(local_input_grad);
  auto&& stream = sync_info.stream_;

  // Get dimensions and strides for each output gradient tensor
  const size_t num_outputs = l.get_num_children();
  std::vector<const TensorDataType*> output_grad_buffer_list;
  std::vector<dim4> output_grad_dims_list, output_grad_strides_list;
  dim4 max_output_grad_dims{0,0,0,0};
  for (size_t j=0; j<num_outputs; ++j) {
    const auto& output_grad = l.get_prev_error_signals(j);
    const auto& output_grad_dims = l.get_output_dims(j);

    // Construct dimensions and strides in reverse order
    // Note: Assume each mini-batch sample is fully packed.
    std::vector<size_t> rdims(output_grad_dims.rbegin(), output_grad_dims.rend());
    std::vector<size_t> rstrides(output_grad_dims.size(), 1);
    for (size_t d=1; d<output_grad_dims.size(); ++d) {
      rstrides[d] = rdims[d-1] * rstrides[d-1];
    }
    rdims.push_back(output_grad.LocalWidth());
    rstrides.push_back(output_grad.LDim());

    // Pad tensor dimensions to 4D
    rdims.resize(4, 1);
    rstrides.resize(4, rstrides.back());

    output_grad_buffer_list.push_back(output_grad.LockedBuffer());
    output_grad_dims_list.push_back({rdims[3], rdims[2], rdims[1], rdims[0]});
    output_grad_strides_list.push_back(
      {rstrides[3], rstrides[2], rstrides[1], rstrides[0]});
    for (size_t i=0; i<4; ++i) {
      max_output_grad_dims[i] = std::max(max_output_grad_dims[i], rdims[3-i]);
    }
  }

  // Get strides for input gradient tensor
  dim4 input_grad_strides;
  {

    // Construct dimensions and strides in reverse order
    // Note: Assume each mini-batch sample is fully packed.
    std::vector<size_t> rdims(input_dims.rbegin(), input_dims.rend());
    std::vector<size_t> rstrides(input_dims.size(), 1);
    for (size_t d=1; d<input_dims.size(); ++d) {
      rstrides[d] = rdims[d-1] * rstrides[d-1];
    }
    rdims.push_back(local_input_grad.Width());
    rstrides.push_back(local_input_grad.LDim());

    // Pad tensor dimensions to 4D
    rdims.resize(4, 1);
    rstrides.resize(4, rstrides.back());

    input_grad_strides = {rstrides[3], rstrides[2], rstrides[1], rstrides[0]};
  }

  // Compute offsets in input gradient tensor
  const size_t slice_dim_stride = input_grad_strides[l.m_slice_dim+(4-num_dims)];
  std::vector<size_t> input_grad_offset_list;
  for (const auto& slice_point : l.m_slice_points) {
    input_grad_offset_list.push_back(slice_point * slice_dim_stride);
  }

  // Pack tensor data into a CPU buffer
  l.m_workspace_event.synchronize();
  l.m_workspace.resize(
    sizeof(TensorDataType*) * output_grad_buffer_list.size()
    + sizeof(dim4) * output_grad_dims_list.size()
    + sizeof(dim4) * output_grad_strides_list.size()
    + sizeof(size_t) * input_grad_offset_list.size());
  size_t pos = 0;
  std::memcpy(&l.m_workspace[pos], output_grad_buffer_list.data(),
              sizeof(TensorDataType*) * output_grad_buffer_list.size());
  pos += sizeof(TensorDataType*) * output_grad_buffer_list.size();
  std::memcpy(&l.m_workspace[pos], output_grad_dims_list.data(),
              sizeof(dim4) * output_grad_dims_list.size());
  pos += sizeof(dim4) * output_grad_dims_list.size();
  std::memcpy(&l.m_workspace[pos], output_grad_strides_list.data(),
              sizeof(dim4) * output_grad_strides_list.size());
  pos += sizeof(dim4) * output_grad_strides_list.size();
  std::memcpy(&l.m_workspace[pos], input_grad_offset_list.data(),
              sizeof(size_t) * input_grad_offset_list.size());
  pos += sizeof(size_t) * input_grad_offset_list.size();

  // Copy tensor data to GPU
  hydrogen::simple_buffer<unsigned char, El::Device::GPU> device_workspace(
    l.m_workspace.size(),
    sync_info);
  unsigned char* device_workspace_ptr = device_workspace.data();
  hipMemcpyAsync(device_workspace_ptr,
                  l.m_workspace.data(),
                  l.m_workspace.size(),
                  hipMemcpyHostToDevice,
                  stream);
  l.m_workspace_event.record(stream);
  pos = 0;
  auto&& device_output_grad_buffer_list
    = reinterpret_cast<const TensorDataType**>(device_workspace_ptr+pos);
  pos += sizeof(TensorDataType*) * output_grad_buffer_list.size();
  auto&& device_output_grad_dims_list
    = reinterpret_cast<const dim4*>(device_workspace_ptr+pos);
  pos += sizeof(dim4) * output_grad_dims_list.size();
  auto&& device_output_grad_strides_list
    = reinterpret_cast<const dim4*>(device_workspace_ptr+pos);
  pos += sizeof(dim4) * output_grad_strides_list.size();
  auto&& device_input_grad_offset_list
    = reinterpret_cast<const size_t*>(device_workspace_ptr+pos);
  pos += sizeof(size_t) * input_grad_offset_list.size();

  // Launch CUDA kernel
  const auto& max_output_grad_size = (max_output_grad_dims[0]
                                      * max_output_grad_dims[1]
                                      * max_output_grad_dims[2]
                                      * max_output_grad_dims[3]);
  if (max_output_grad_size > 0) {
    constexpr size_t block_size = 64;
    dim3 block_dims, grid_dims;
    block_dims.x = block_size;
    grid_dims.x = (max_output_grad_dims[3] + block_size - 1) / block_size;
    grid_dims.y = max_output_grad_dims[2];
    grid_dims.z = max_output_grad_dims[1];
    concat4d_kernel<<<grid_dims, block_dims, 0, stream>>>(
      num_outputs,
      device_output_grad_buffer_list,
      device_output_grad_dims_list,
      device_output_grad_strides_list,
      local_input_grad.Buffer(),
      input_grad_strides,
      device_input_grad_offset_list);
  }

}

// Explicit instantiation
#define PROTO(T)                                        \
  template class slice_layer<                           \
    T, data_layout::DATA_PARALLEL, El::Device::GPU>;    \
  template class slice_layer<                           \
    T, data_layout::MODEL_PARALLEL, El::Device::GPU>

#define LBANN_INSTANTIATE_GPU_HALF
#include "lbann/macros/instantiate.hpp"

} // namespace lbann
