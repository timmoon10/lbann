#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// Copyright (c) 2014-2019, Lawrence Livermore National Security, LLC.
// Produced at the Lawrence Livermore National Laboratory.
// Written by the LBANN Research Team (B. Van Essen, et al.) listed in
// the CONTRIBUTORS file. <lbann-dev@llnl.gov>
//
// LLNL-CODE-697807.
// All rights reserved.
//
// This file is part of LBANN: Livermore Big Artificial Neural Network
// Toolkit. For details, see http://software.llnl.gov/LBANN or
// https://github.com/LLNL/LBANN.
//
// Licensed under the Apache License, Version 2.0 (the "Licensee"); you
// may not use this file except in compliance with the License.  You may
// obtain a copy of the License at:
//
// http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or
// implied. See the License for the specific language governing
// permissions and limitations under the license.
////////////////////////////////////////////////////////////////////////////////

#define LBANN_SOFTMAX_LAYER_INSTANTIATE
#include "lbann/layers/activations/softmax.hpp"
#include "lbann/utils/cuda.hpp"

namespace lbann {

namespace {

#ifdef LBANN_ENABLE_SOFTMAX_THRESHOLD
/** Functor to ensure values are above threshold value */
template <typename TensorDataType>
struct threshold_op {
  __forceinline__ __device__ TensorDataType operator()(const TensorDataType& y) const {
    return cuda::max(y, cuda::sqrt(cuda::min<TensorDataType>()));
  }
};
#endif // LBANN_ENABLE_SOFTMAX_THRESHOLD

/** @brief Max functor */
template <class T>
struct max_op {
  __device__ __forceinline__
  DataType operator()(const T& x1, const T& x2) const {
    return cuda::max(x1, x2);
  }
};

/** @brief Kernel for max reduction on matrix columns
 *
 *  Each CUDA block computes the max over a subset of matrix entries
 *  and outputs the result. This is repeated multiple times for
 *  column-wise max reduction.
 *
 *  Block dimensions: bsize x 1 x 1
 *
 *  Grid dimension: (height / bsize) x width x 1
 *
 *  @param values       (height x width) matrix
 *  @param max_values   (nblocksx x width) matrix
 */
template <size_t bsize, typename TensorDataType>
__global__ void reduce_max_kernel(size_t height,
                                  size_t width,
                                  const TensorDataType* __restrict__ values,
                                  size_t values_ldim,
                                  TensorDataType* __restrict__ max_values) {

  // Indices
  const size_t tid = threadIdx.x;
  const size_t gidx = threadIdx.x + blockIdx.x * blockDim.x;
  const size_t bidx = blockIdx.x;
  const size_t bidy = blockIdx.y;
  const size_t nthreadsx = blockDim.x * gridDim.x;
  const size_t nblocksx = gridDim.x;
  const size_t nblocksy = gridDim.y;

  for (size_t col = bidy; col < width; col += nblocksy) {

    // Find largest value for each thread
    TensorDataType thread_max_val{-cuda::infinity<DataType>()};
    for (size_t row = gidx; row < height; row += nthreadsx) {
      const auto& val = values[row+col*values_ldim];
      thread_max_val = cuda::max(thread_max_val, val);
    }

    // Find largest value for each block
    const TensorDataType block_max_val
      = cuda::block_reduce<bsize,1,1,DataType,max_op<DataType>>(thread_max_val);
    if (tid == 0) {
      max_values[bidx+col*nblocksx] = block_max_val;
    }

  }

}

/** @brief Compute exp(x-shift)
 *
 *  Also compute sum(exp(x-shift)) for each matrix column.
 *
 *  Block dimensions: bsize x 1 x 1
 *
 *  Grid dimension: (height / bsize) x width x 1
 */
template <size_t bsize, typename TensorDataType>
__global__ void fp_exp_kernel(size_t height,
                              size_t width,
                              const TensorDataType* __restrict__ input,
                              size_t input_ldim,
                              TensorDataType* __restrict__ output,
                              size_t output_ldim,
                              const TensorDataType* __restrict__ shifts,
                              TensorDataType* __restrict__ sums) {

  // Indices
  const size_t tid = threadIdx.x;
  const size_t gidx = threadIdx.x + blockIdx.x * blockDim.x;
  const size_t bidy = blockIdx.y;
  const size_t nthreadsx = blockDim.x * gridDim.x;
  const size_t nblocksy = gridDim.y;

  for (size_t col = bidy; col < width; col += nblocksy) {
    const auto& shift = shifts[col];

    // Exponentiate inputs and compute sum for each thread
    TensorDataType thread_sum{0};
    for (size_t row = gidx; row < height; row += nthreadsx) {
      const auto& x = input[row+col*input_ldim];
      auto& y = output[row+col*output_ldim];
      y = cuda::exp(x-shift);
      thread_sum += y;
    }

    // Compute sum for each block
    const TensorDataType block_sum = cuda::block_reduce<bsize,1,1>(thread_sum);
    if (tid == 0) {
      cuda::atomic_add(&sums[col], block_sum);
    }

  }

}

/** @brief Compute layer output
 *
 *  y = exp(x-shift) / sum(exp(x-shift))
 *
 *  If @c LBANN_ENABLE_SOFTMAX_THRESHOLD is set, small values are
 *  thresholded to a minimum value to avoid denormalized floats.
 *
 *  Block dimensions: bsize x 1 x 1
 *
 *  Grid dimension: (height / bsize) x width x 1
 *
 *  @param output   On input, constains exp(x-shift). On output,
 *                  contains the layer output.
 *  @param sums     sum(exp(x-shift)) for each column
 */
template <typename TensorDataType>
__global__ void fp_output_kernel(size_t height,
                                 size_t width,
                                 TensorDataType* __restrict__ output,
                                 size_t output_ldim,
                                 const TensorDataType* __restrict__ sums) {
  const size_t gidx = threadIdx.x + blockIdx.x * blockDim.x;
  const size_t gidy = threadIdx.y + blockIdx.y * blockDim.y;
  const size_t nthreadsx = blockDim.x * gridDim.x;
  const size_t nthreadsy = blockDim.y * gridDim.y;
  for (size_t col = gidy; col < width; col += nthreadsy) {
    const auto& denom = sums[col];
    for (size_t row = gidx; row < height; row += nthreadsx) {
      auto& y = output[row+col*output_ldim];
      y /= denom;
#ifdef LBANN_ENABLE_SOFTMAX_THRESHOLD
      y = cuda::max(y, cuda::sqrt(cuda::min<TensorDataType>()));
#endif // LBANN_ENABLE_SOFTMAX_THRESHOLD
    }
  }
}

/** @brief Compute dot(y,dy) for each matrix column
 *
 *  Block dimensions: bsize x 1 x 1
 *
 *  Grid dimension: (height / bsize) x width x 1
 */
template <size_t bsize, typename TensorDataType>
__global__ void bp_dot_product_kernel(
  size_t height,
  size_t width,
  const TensorDataType* __restrict__ output,
  size_t output_ldim,
  const TensorDataType* __restrict__ gradient_wrt_output,
  size_t gradient_wrt_output_ldim,
  TensorDataType* __restrict__ dot_products) {

  // Indices
  const size_t tid = threadIdx.x;
  const size_t gidx = threadIdx.x + blockIdx.x * blockDim.x;
  const size_t bidy = blockIdx.y;
  const size_t nthreadsx = blockDim.x * gridDim.x;
  const size_t nblocksy = gridDim.y;

  for (size_t col = bidy; col < width; col += nblocksy) {

    // Compute dot product contribution for each thread
    TensorDataType thread_dot_product{0};
    for (size_t row = gidx; row < height; row += nthreadsx) {
      const auto& y = output[row+col*output_ldim];
      const auto& dy = gradient_wrt_output[row+col*gradient_wrt_output_ldim];
      thread_dot_product += y * dy;
    }

    // Compute dot product contribution for each block
    const TensorDataType block_dot_product
      = cuda::block_reduce<bsize,1,1>(thread_dot_product);
    if (tid == 0) {
      cuda::atomic_add(&dot_products[col], block_dot_product);
    }

  }

}

/** @brief Compute gradient w.r.t. input
 *
 *  dx = y * (dy - dot(y,dy))
 *
 *  Block dimensions: bsize x 1 x 1
 *
 *  Grid dimension: (height / bsize) x width x 1
 *
 *  @param dot_products dot(y,dy) for each matrix column
 */
template <size_t bsize, typename TensorDataType>
__global__ void bp_kernel(size_t height,
                          size_t width,
                          const TensorDataType* __restrict__ output,
                          size_t output_ldim,
                          const TensorDataType* __restrict__ gradient_wrt_output,
                          size_t gradient_wrt_output_ldim,
                          const TensorDataType* __restrict__ dot_products,
                          TensorDataType* __restrict__ gradient_wrt_input,
                          size_t gradient_wrt_input_ldim) {
  const size_t gidx = threadIdx.x + blockIdx.x * blockDim.x;
  const size_t gidy = threadIdx.y + blockIdx.y * blockDim.y;
  const size_t nthreadsx = blockDim.x * gridDim.x;
  const size_t nthreadsy = blockDim.y * gridDim.y;
  for (size_t col = gidy; col < width; col += nthreadsy) {
    const auto& y_dot_dy = dot_products[col];
    for (size_t row = gidx; row < height; row += nthreadsx) {
      const auto& y = output[row+col*output_ldim];
      const auto& dy = gradient_wrt_output[row+col*gradient_wrt_output_ldim];
      auto& dx = gradient_wrt_input[row+col*gradient_wrt_input_ldim];
      dx = y * (dy - y_dot_dy);
    }
  }
}

#ifdef LBANN_HAS_DISTCONV
template <typename TensorDataType, data_layout Layout, El::Device Device>
void fp_compute_distconv(softmax_distconv_adapter<TensorDataType, Layout, Device> &dc) {
  dc.m_softmax->forward(dc.get_prev_activations(), dc.get_activations());
}

template <typename TensorDataType, data_layout Layout, El::Device Device>
void bp_compute_distconv(softmax_distconv_adapter<TensorDataType, Layout, Device> &dc) {
  dc.m_softmax->backward(dc.get_activations(),
                         dc.get_prev_error_signals(),
                         dc.get_error_signals());
}
#endif // LBANN_HAS_DISTCONV

} // namespace

template <typename TensorDataType>
void fp_compute_impl(softmax_layer<TensorDataType, data_layout::DATA_PARALLEL, El::Device::GPU>& l) {
#ifdef LBANN_HAS_DISTCONV
  if (l.distconv_enabled()) {
    fp_compute_distconv(l.get_distconv_adapter());
    return;
  }
#endif // LBANN_HAS_DISTCONV

  hipdnnSoftmaxMode_t cudnn_softmax_mode;
  switch(l.m_mode) {
    case softmax_mode::INSTANCE:
      cudnn_softmax_mode = HIPDNN_SOFTMAX_MODE_INSTANCE;
      break;
    case softmax_mode::CHANNEL:
      cudnn_softmax_mode = HIPDNN_SOFTMAX_MODE_CHANNEL;
      break;
    default:
      LBANN_ERROR("Unsupported softmax mode");
  }

  const cudnn::ScalingParamType<TensorDataType> zero = 0.;
  const cudnn::ScalingParamType<TensorDataType> one = 1.;
  const auto& local_input = dynamic_cast<const El::Matrix<TensorDataType, El::Device::GPU>&>(l.get_local_prev_activations());
  auto& local_output = dynamic_cast<El::Matrix<TensorDataType, El::Device::GPU>&>(l.get_local_activations());
  if (!local_input.IsEmpty()) {
    CHECK_CUDNN(hipdnnSoftmaxForward(cudnn::get_handle(),
                                    HIPDNN_SOFTMAX_ACCURATE,
                                    cudnn_softmax_mode,
                                    &one,
                                    l.m_tensors_cudnn_desc.get_prev_activations(),
                                    local_input.LockedBuffer(),
                                    &zero,
                                    l.m_tensors_cudnn_desc.get_activations(),
                                    local_output.Buffer()));
#ifdef LBANN_ENABLE_SOFTMAX_THRESHOLD
    cuda::apply_entrywise_unary_operator<threshold_op>(local_output,
                                                       local_output);
#endif // LBANN_ENABLE_SOFTMAX_THRESHOLD
  }
}

template <typename TensorDataType>
void bp_compute_impl(softmax_layer<TensorDataType, data_layout::DATA_PARALLEL, El::Device::GPU>& l) {
#ifdef LBANN_HAS_DISTCONV
  if (l.distconv_enabled()) {
    bp_compute_distconv(l.get_distconv_adapter());
    return;
  }
#endif // LBANN_HAS_DISTCONV

  hipdnnSoftmaxMode_t cudnn_softmax_mode;
  switch(l.m_mode) {
    case softmax_mode::INSTANCE:
      cudnn_softmax_mode = HIPDNN_SOFTMAX_MODE_INSTANCE;
      break;
    case softmax_mode::CHANNEL:
      cudnn_softmax_mode = HIPDNN_SOFTMAX_MODE_CHANNEL;
      break;
    default:
      LBANN_ERROR("Unsupported softmax mode");
  }

  const cudnn::ScalingParamType<TensorDataType> zero = 0.;
  const cudnn::ScalingParamType<TensorDataType> one = 1.;
  const auto& local_output = dynamic_cast<const El::Matrix<TensorDataType, El::Device::GPU>&>(l.get_local_activations());
  const auto& local_gradient_wrt_output = dynamic_cast<const El::Matrix<TensorDataType, El::Device::GPU>&>(l.get_local_prev_error_signals());
  auto& local_gradient_wrt_input = dynamic_cast<El::Matrix<TensorDataType, El::Device::GPU>&>(l.get_local_error_signals());
  if (!local_output.IsEmpty()) {
    CHECK_CUDNN(hipdnnSoftmaxBackward(cudnn::get_handle(),
                                     HIPDNN_SOFTMAX_ACCURATE,
                                     cudnn_softmax_mode,
                                     &one,
                                     l.m_tensors_cudnn_desc.get_activations(),
                                     local_output.LockedBuffer(),
                                     l.m_tensors_cudnn_desc.get_prev_error_signals(),
                                     local_gradient_wrt_output.LockedBuffer(),
                                     &zero,
                                     l.m_tensors_cudnn_desc.get_error_signals(),
                                     local_gradient_wrt_input.Buffer()));
  }
}

template <typename TensorDataType>
void fp_compute_impl(softmax_layer<TensorDataType, data_layout::MODEL_PARALLEL, El::Device::GPU>& l) {

  if(l.m_mode != softmax_mode::INSTANCE) {
    LBANN_ERROR("Unsupported softmax mode");
  }

  // Local matrices
  const auto& local_input = dynamic_cast<const El::Matrix<TensorDataType, El::Device::GPU>&>(l.get_local_prev_activations());
  auto& local_output = dynamic_cast<El::Matrix<TensorDataType, El::Device::GPU>&>(l.get_local_activations());
  auto& local_workspace = dynamic_cast<El::Matrix<TensorDataType, El::Device::GPU>&>(l.m_workspace->Matrix());
  const size_t local_height = local_input.Height();
  const size_t local_width = local_input.Width();

  // GPU objects
  auto&& stream = El::GPUManager::Stream();
  auto&& event = El::GPUManager::Event();
  El::SyncInfo<El::Device::GPU> sync_info{stream, event};

  // Find max value in each column
  cuda::thrust::vector<TensorDataType> max_vals;
  if (local_output.IsEmpty()) {
    max_vals.resize(local_width,
                    -std::numeric_limits<TensorDataType>::infinity());
  }
  else {
    constexpr size_t block_size = 256;
    dim3 block_dims, grid_dims;
    block_dims.x = block_size;
    grid_dims.x = (local_height + block_size - 1) / block_size;
    grid_dims.y = local_width;
    max_vals.resize(grid_dims.x * local_width);
    reduce_max_kernel<block_size><<<grid_dims, block_dims, 0, stream>>>(
      local_height, local_width,
      local_input.LockedBuffer(), local_input.LDim(),
      max_vals.data().get());
    while (grid_dims.x > 1) {
      const size_t prev_height = grid_dims.x;
      grid_dims.x = (prev_height + block_size - 1) / block_size;
      cuda::thrust::vector<TensorDataType> prev_vals(std::move(max_vals));
      max_vals.resize(grid_dims.x * local_width);
      reduce_max_kernel<block_size><<<grid_dims, block_dims, 0, stream>>>(
        prev_height, local_width,
        prev_vals.data().get(), prev_height,
        max_vals.data().get());
    }
  }
  El::mpi::AllReduce(max_vals.data().get(), max_vals.size(),
                     El::mpi::MAX, l.m_workspace->RedundantComm(),
                     sync_info);

  // Compute exp(x-max_val) and sum(exp(x-max_val))
  El::Zero(*l.m_workspace);
  if (!local_output.IsEmpty()) {
    constexpr size_t block_size = 256;
    dim3 block_dims, grid_dims;
    block_dims.x = block_size;
    grid_dims.x = (local_height + block_size - 1) / block_size;
    grid_dims.y = local_width;
    fp_exp_kernel<block_size><<<grid_dims, block_dims, 0, stream>>>(
      local_height, local_width,
      local_input.LockedBuffer(), local_input.LDim(),
      local_output.Buffer(), local_output.LDim(),
      max_vals.data().get(),
      local_workspace.Buffer());
  }
  El::AllReduce(*l.m_workspace, l.m_workspace->RedundantComm());

  // Compute output
  // Note: y = exp(x-max_val) / sum(exp(x-max_val))
  if (!local_output.IsEmpty()) {
    constexpr size_t block_size = 256;
    dim3 block_dims, grid_dims;
    block_dims.x = block_size;
    grid_dims.x = (local_height + block_size - 1) / block_size;
    grid_dims.y = local_width;
    fp_output_kernel<<<grid_dims, block_dims, 0, stream>>>(
      local_height, local_width,
      local_output.Buffer(), local_output.LDim(),
      local_workspace.LockedBuffer());
  }

}

template <typename TensorDataType>
void bp_compute_impl(softmax_layer<TensorDataType, data_layout::MODEL_PARALLEL, El::Device::GPU>& l) {

  if(l.m_mode != softmax_mode::INSTANCE) {
    LBANN_ERROR("Unsupported softmax mode");
  }

  // Local matrices
  const auto& local_output = dynamic_cast<const El::Matrix<TensorDataType, El::Device::GPU>&>(l.get_local_activations());
  const auto& local_gradient_wrt_output = dynamic_cast<const El::Matrix<TensorDataType, El::Device::GPU>&>(l.get_local_prev_error_signals());
  auto& local_gradient_wrt_input = dynamic_cast<El::Matrix<TensorDataType, El::Device::GPU>&>(l.get_local_error_signals());
  auto& local_workspace = dynamic_cast<El::Matrix<TensorDataType, El::Device::GPU>&>(l.m_workspace->Matrix());
  const auto& local_height = local_output.Height();
  const auto& local_width = local_output.Width();

  // GPU objects
  auto&& stream = El::GPUManager::Stream();
  auto&& event = El::GPUManager::Event();
  El::SyncInfo<El::Device::GPU> sync_info{stream, event};

  // Compute dot(y,dy)
  El::Zero(local_workspace);
  if (!local_output.IsEmpty()) {
    constexpr size_t block_size = 256;
    dim3 block_dims, grid_dims;
    block_dims.x = block_size;
    grid_dims.x = (local_height + block_size - 1) / block_size;
    grid_dims.y = local_width;
    bp_dot_product_kernel<block_size>
      <<<grid_dims, block_dims, 0, stream>>>(
        local_height, local_width,
        local_output.LockedBuffer(),
        local_output.LDim(),
        local_gradient_wrt_output.LockedBuffer(),
        local_gradient_wrt_output.LDim(),
        local_workspace.Buffer());
  }
  El::AllReduce(*l.m_workspace, l.m_workspace->RedundantComm());

  // Compute gradient w.r.t. input
  if (!local_output.IsEmpty()) {
    constexpr size_t block_size = 256;
    dim3 block_dims, grid_dims;
    block_dims.x = block_size;
    grid_dims.x = (local_height + block_size - 1) / block_size;
    grid_dims.y = local_width;
    bp_kernel<block_size><<<grid_dims, block_dims, 0, stream>>>(
      local_height, local_width,
      local_output.LockedBuffer(),
      local_output.LDim(),
      local_gradient_wrt_output.LockedBuffer(),
      local_gradient_wrt_output.LDim(),
      local_workspace.Buffer(),
      local_gradient_wrt_input.Buffer(),
      local_gradient_wrt_input.LDim());
  }

}

template <typename TensorDataType, data_layout Layout, El::Device Device>
void softmax_layer<TensorDataType, Layout, Device>::fp_compute() {
  fp_compute_impl(*this);
}
template <typename TensorDataType, data_layout Layout, El::Device Device>
void softmax_layer<TensorDataType, Layout, Device>::bp_compute() {
  bp_compute_impl(*this);
}

// Template instantiation
#define PROTO(T)                                      \
  template class softmax_layer<T, data_layout::DATA_PARALLEL, El::Device::GPU>; \
  template class softmax_layer<T, data_layout::MODEL_PARALLEL, El::Device::GPU>

#define LBANN_INSTANTIATE_GPU_HALF
#include "lbann/macros/instantiate.hpp"

} // namespace lbann
