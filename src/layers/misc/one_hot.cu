#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// Copyright (c) 2014-2019, Lawrence Livermore National Security, LLC.
// Produced at the Lawrence Livermore National Laboratory.
// Written by the LBANN Research Team (B. Van Essen, et al.) listed in
// the CONTRIBUTORS file. <lbann-dev@llnl.gov>
//
// LLNL-CODE-697807.
// All rights reserved.
//
// This file is part of LBANN: Livermore Big Artificial Neural Network
// Toolkit. For details, see http://software.llnl.gov/LBANN or
// https://github.com/LLNL/LBANN.
//
// Licensed under the Apache License, Version 2.0 (the "Licensee"); you
// may not use this file except in compliance with the License.  You may
// obtain a copy of the License at:
//
// http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or
// implied. See the License for the specific language governing
// permissions and limitations under the license.
////////////////////////////////////////////////////////////////////////////////

#define LBANN_ONE_HOT_LAYER_INSTANTIATE
#include "lbann/layers/misc/one_hot.hpp"

namespace lbann {

namespace {

/**
 *  On input, output is assumed to be filled with zeros.
 *
 *  Block dimensions: bsize x 1 x 1
 *
 *  Grid dimensions: (width / bsize) x 1 x 1
 */
template <typename TensorDataType>
__global__ void fp_kernel(unsigned long long height,
                          unsigned long long width,
                          const TensorDataType* __restrict__ indices,
                          unsigned long long indices_stride,
                          TensorDataType* __restrict__ output,
                          unsigned long long output_ldim) {
  const unsigned long long gid = threadIdx.x + blockIdx.x * blockDim.x;
  const unsigned long long nthreads = blockDim.x * gridDim.x;
  for (unsigned long long col = gid; col < width; col += nthreads) {
    const auto& ind = indices[col*indices_stride];
    if (TensorDataType(0.f) <= ind && ind < TensorDataType(height)) {
      const unsigned long long row = static_cast<unsigned long long>(ind);
      output[row+col*output_ldim] = TensorDataType(1.f);
    }
  }
}

} // namespace <anon>

template <typename TensorDataType, data_layout Layout, El::Device Device>
void one_hot_layer<TensorDataType, Layout, Device>::fp_compute() {

  using GPUMatType = El::Matrix<TensorDataType, El::Device::GPU>;

  // Local matrices
  const auto& local_input =
    dynamic_cast<const GPUMatType&>(this->get_local_prev_activations());
  auto& local_output = dynamic_cast<GPUMatType&>(this->get_local_activations());

  // Populate one-hot vectors
  El::Zero(local_output);
  if (!local_output.IsEmpty()) {
    const size_t local_height = local_output.Height();
    const size_t local_width = local_output.Width();
    constexpr size_t block_size = 64;
    const size_t grid_size = (local_width + block_size - 1) / block_size;
    fp_kernel<<<grid_size, block_size, 0, El::GPUManager::Stream()>>>(
        local_height,
        local_width,
        local_input.LockedBuffer(),
        local_input.LDim(),
        local_output.Buffer(),
        local_output.LDim());
  }

}

#define PROTO(T)                     \
  template class one_hot_layer<T, data_layout::DATA_PARALLEL, El::Device::GPU>

#define LBANN_INSTANTIATE_GPU_HALF
#include "lbann/macros/instantiate.hpp"

} // namespace lbann
