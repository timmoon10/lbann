#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// Copyright (c) 2014-2019, Lawrence Livermore National Security, LLC.
// Produced at the Lawrence Livermore National Laboratory.
// Written by the LBANN Research Team (B. Van Essen, et al.) listed in
// the CONTRIBUTORS file. <lbann-dev@llnl.gov>
//
// LLNL-CODE-697807.
// All rights reserved.
//
// This file is part of LBANN: Livermore Big Artificial Neural Network
// Toolkit. For details, see http://software.llnl.gov/LBANN or
// https://github.com/LLNL/LBANN.
//
// Licensed under the Apache License, Version 2.0 (the "Licensee"); you
// may not use this file except in compliance with the License.  You may
// obtain a copy of the License at:
//
// http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or
// implied. See the License for the specific language governing
// permissions and limitations under the license.
////////////////////////////////////////////////////////////////////////////////

#define LBANN_ENTRYWISE_BATCH_NORMALIZATION_LAYER_INSTANTIATE
#include "lbann/layers/regularizers/entrywise_batch_normalization.hpp"
#include "lbann/utils/cuda.hpp"

namespace lbann {

namespace {

/**
 *  On input, sums and sqsums are assumed to be filled with zeros.
 *
 *  Block dimensions: bsize x 1 x 1
 *
 *  Grid dimensions: (height / bsize) x 1 x 1
 */
template <typename TensorDataType>
__global__ void row_sums_kernel(size_t height,
                                size_t width,
                                const TensorDataType* __restrict__ vals,
                                size_t vals_ldim,
                                TensorDataType* __restrict__ sums,
                                TensorDataType* __restrict__ sqsums) {
  const size_t gid = threadIdx.x + blockIdx.x * blockDim.x;
  const size_t nthreads = blockDim.x * gridDim.x;
  for (size_t row = gid; row < height; row += nthreads) {
    auto& sum = sums[row];
    auto& sqsum = sqsums[row];
    for (size_t col = 0; col < width; ++col) {
      const auto& x = vals[row + col * vals_ldim];
      sum += x;
      sqsum += x * x;
    }
  }
}

/**
 *  On input, batch_mean and batch_var are assumed to contain sums and
 *  squares of sums, respectively.
 *
 *  Block dimensions: bsize x 1 x 1
 *
 *  Grid dimensions: (size / bsize) x 1 x 1
 */
template <typename TensorDataType>
__global__ void compute_statistics_kernel(size_t size,
                                          unsigned long long statistics_count,
                                          TensorDataType decay,
                                          TensorDataType* __restrict__ batch_mean,
                                          TensorDataType* __restrict__ batch_var,
                                          TensorDataType* __restrict__ running_mean,
                                          TensorDataType* __restrict__ running_var) {
  const size_t gid = threadIdx.x + blockIdx.x * blockDim.x;
  const size_t nthreads = blockDim.x * gridDim.x;
  for (size_t i = gid; i < size; i += nthreads) {
    auto& mean = batch_mean[i];
    auto& var = batch_var[i];
    auto& _running_mean = running_mean[i];
    auto& _running_var = running_var[i];
    const auto sum = batch_mean[i];
    const auto sqsum = batch_var[i];
    const TensorDataType statistics_count_dt = TensorDataType(statistics_count);
    mean = sum / statistics_count_dt;
    const auto sqmean = sqsum / statistics_count_dt;
    var = (sqmean - mean * mean) * statistics_count_dt / TensorDataType(statistics_count - 1);
    _running_mean = decay * _running_mean + (TensorDataType{1} - decay) * mean;
    _running_var = decay * _running_var + (TensorDataType{1} - decay) * var;
  }
}

/**
 *  mean = sum(x_i) / n
 *
 *  var = ( sum(x_i^2)/n - mean^2 ) * n/(n-1)
 */
template <typename TensorDataType>
void compute_batch_statistics(lbann_comm& comm,
                              TensorDataType decay,
                              const El::AbstractDistMatrix<TensorDataType>& input,
                              El::AbstractDistMatrix<TensorDataType>& batch_statistics,
                              El::AbstractDistMatrix<TensorDataType>& running_mean,
                              El::AbstractDistMatrix<TensorDataType>& running_var) {

  // Local matrices
  const auto& local_input = dynamic_cast<const El::Matrix<TensorDataType, El::Device::GPU>&>(input.LockedMatrix());
  auto& local_batch_statistics = dynamic_cast<El::Matrix<TensorDataType, El::Device::GPU>&>(batch_statistics.Matrix());
  auto local_batch_mean = El::View(local_batch_statistics, El::ALL, El::IR(0));
  auto local_batch_var = El::View(local_batch_statistics, El::ALL, El::IR(1));
  auto& local_running_mean = dynamic_cast<El::Matrix<TensorDataType, El::Device::GPU>&>(running_mean.Matrix());
  auto& local_running_var = dynamic_cast<El::Matrix<TensorDataType, El::Device::GPU>&>(running_var.Matrix());

  // Dimensions
  const size_t local_height = local_input.Height();
  const size_t local_width = local_input.Width();

  // Compute local sums
  El::Zero(batch_statistics);
  if (local_height > 0) {
    constexpr size_t block_size = 256;
    dim3 block_dims, grid_dims;
    block_dims.x = block_size;
    grid_dims.x = (local_height + block_size - 1) / block_size;
    row_sums_kernel<TensorDataType>
      <<<grid_dims, block_dims, 0, El::GPUManager::Stream()>>>(
        local_height,
        local_width,
        local_input.LockedBuffer(),
        local_input.LDim(),
        local_batch_mean.Buffer(),
        local_batch_var.Buffer());
  }

  // Accumulate sums between processes
  /// @todo Local statistics
  /// @todo Arbitrary group sizes
  comm.allreduce(batch_statistics,
                 batch_statistics.RedundantComm(),
                 El::mpi::SUM);
  const size_t statistics_count = input.Width();

  // Compute mini-batch statistics from sums
  if (statistics_count <= 1) {
    // local_mean already has correct values
    El::Fill(local_batch_var, El::TypeTraits<TensorDataType>::One());
  } else {
    if (local_height > 0) {
      constexpr size_t block_size = 256;
      dim3 block_dims, grid_dims;
      block_dims.x = block_size;
      grid_dims.x = (local_height + block_size - 1) / block_size;
      compute_statistics_kernel<TensorDataType>
        <<<grid_dims, block_dims, 0, El::GPUManager::Stream()>>>(
          local_height,
          statistics_count,
          decay,
          local_batch_mean.Buffer(),
          local_batch_var.Buffer(),
          local_running_mean.Buffer(),
          local_running_var.Buffer());
    }
  }

}

/**
 *  Block dimensions: bsizex x bsizey x 1
 *
 *  Grid dimensions: (height / bsizex) x (width / bsizey) x 1
 */
template <typename TensorDataType>
__global__ void batchnorm_kernel(size_t height,
                                 size_t width,
                                 TensorDataType epsilon,
                                 const TensorDataType* __restrict__ input,
                                 size_t input_ldim,
                                 TensorDataType* __restrict__ output,
                                 size_t output_ldim,
                                 const TensorDataType* __restrict__ mean,
                                 const TensorDataType* __restrict__ var) {
  const size_t gidx = threadIdx.x + blockIdx.x * blockDim.x;
  const size_t gidy = threadIdx.y + blockIdx.y * blockDim.y;
  const size_t nthreadsx = blockDim.x * gridDim.x;
  const size_t nthreadsy = blockDim.y * gridDim.y;
  for (size_t row = gidx; row < height; row += nthreadsx) {
    const auto& _mean = mean[row];
    const auto& _var = var[row];
    const auto inv_stdev = cuda::rsqrt(_var + epsilon);
    for (size_t col = gidy; col < width; col += nthreadsy) {
      const auto& x = input[row + col*input_ldim];
      auto& y = output[row + col*output_ldim];
      y = (x - _mean) * inv_stdev;
    }
  }
}

/**
 *  y_i = (x_i - mean) / sqrt(var + epsilon)
 */
template <typename TensorDataType>
void apply_batchnorm(DataType epsilon,
                     const El::Matrix<TensorDataType, El::Device::GPU>& local_input,
                     El::Matrix<TensorDataType, El::Device::GPU>& local_output,
                     const El::Matrix<TensorDataType, El::Device::GPU>& local_mean,
                     const El::Matrix<TensorDataType, El::Device::GPU>& local_var) {
  if (!local_input.IsEmpty()) {
    const size_t local_height = local_input.Height();
    const size_t local_width = local_input.Width();
    constexpr size_t block_size_x = 256;
    constexpr size_t block_size_y = 1;
    dim3 block_dims, grid_dims;
    block_dims.x = block_size_x;
    block_dims.y = block_size_y;
    grid_dims.x = (local_height + block_size_x - 1) / block_size_x;
    grid_dims.y = (local_width + block_size_y - 1) / block_size_y;
    batchnorm_kernel<TensorDataType>
      <<<grid_dims, block_dims, 0, El::GPUManager::Stream()>>>(
        local_height,
        local_width,
        epsilon,
        local_input.LockedBuffer(),
        local_input.LDim(),
        local_output.Buffer(),
        local_output.LDim(),
        local_mean.LockedBuffer(),
        local_var.LockedBuffer());
  }
}

template <typename TensorDataType>
void fp_impl(lbann_comm& comm,
             TensorDataType decay,
             TensorDataType epsilon,
             bool is_training,
             const El::AbstractDistMatrix<TensorDataType>& input,
             El::AbstractDistMatrix<TensorDataType>& output,
             El::AbstractDistMatrix<TensorDataType>& batch_statistics,
             El::AbstractDistMatrix<TensorDataType>& running_mean,
             El::AbstractDistMatrix<TensorDataType>& running_var) {

  // Local matrices
  const auto& local_input = dynamic_cast<const El::Matrix<TensorDataType, El::Device::GPU>&>(input.LockedMatrix());
  auto& local_output = dynamic_cast<El::Matrix<TensorDataType, El::Device::GPU>&>(output.Matrix());

  // Batchnorm has different behavior for training and inference
  if (is_training) {

    // For training, normalize with batch statistics
    compute_batch_statistics<TensorDataType>(comm,
                                             decay,
                                             input,
                                             batch_statistics,
                                             running_mean,
                                             running_var);
    const auto& local_batch_statistics
      = dynamic_cast<const El::Matrix<TensorDataType, El::Device::GPU>&>(batch_statistics.LockedMatrix());
    const auto local_batch_mean = El::LockedView(local_batch_statistics,
                                                 El::ALL, El::IR(0));
    const auto local_batch_var = El::LockedView(local_batch_statistics,
                                                El::ALL, El::IR(1));
    apply_batchnorm<TensorDataType>(epsilon,
                                    local_input,
                                    local_output,
                                    local_batch_mean,
                                    local_batch_var);

  }
  else {

    // For inference, normalize with running statistics
    const auto& local_running_mean = dynamic_cast<const El::Matrix<TensorDataType, El::Device::GPU>&>(running_mean.LockedMatrix());
    const auto& local_running_var = dynamic_cast<const El::Matrix<TensorDataType, El::Device::GPU>&>(running_var.LockedMatrix());
    apply_batchnorm<TensorDataType>(epsilon,
                                    local_input,
                                    local_output,
                                    local_running_mean,
                                    local_running_var);

  }

}

/**
 *  On input, gradient_wrt_mean and gradient_wrt_var are assumed to be
 *  filled with zeros.
 *
 *  dL/dmean = - sum(dL/dy_i) / sqrt(var+epsilon)
 *
 *  dL/dvar = - sum(dL/dy_i * (x_i-mean)) * (var+epsilon)^(-3/2) / 2
 *
 *  Block dimensions: bsize x 1 x 1
 *
 *  Grid dimensions: (height / bsize) x 1 x 1
 */
template <typename TensorDataType>
__global__ void bp_training_stats_gradient_kernel(size_t height,
                                                  size_t width,
                                                  TensorDataType epsilon,
                                                  const TensorDataType* __restrict__ input,
                                                  size_t input_ldim,
                                                  const TensorDataType* __restrict__ gradient_wrt_output,
                                                  size_t gradient_wrt_output_ldim,
                                                  const TensorDataType* __restrict__ mean,
                                                  const TensorDataType* __restrict__ var,
                                                  TensorDataType* __restrict__ gradient_wrt_mean,
                                                  TensorDataType* __restrict__ gradient_wrt_var) {
  const size_t gid = threadIdx.x + blockIdx.x * blockDim.x;
  const size_t nthreads = blockDim.x * gridDim.x;
  for (size_t row = gid; row < height; row += nthreads) {
    const auto& _mean = mean[row];
    const auto& _var = var[row];
    const auto inv_stdev = cuda::rsqrt(_var + epsilon);
    auto& dmean = gradient_wrt_mean[row];
    auto& dvar = gradient_wrt_var[row];
    for (size_t col = 0; col < width; ++col) {
      const auto& x = input[row + col * input_ldim];
      const auto& dy = gradient_wrt_output[row + col * gradient_wrt_output_ldim];
      dmean += - dy * inv_stdev;
      dvar += - dy * (x - _mean) * inv_stdev*inv_stdev*inv_stdev / TensorDataType(2);
    }
  }
}

/**
 *  dL/dx_i = ( dL/dy_i / sqrt(var+epsilon)
 *              + dL/dmean / n
 *              + dL/dvar * (x_i - mean) * 2/(n-1) )
 *
 *  Block dimensions: bsizex x bsizey x 1
 *
 *  Grid dimensions: (height / bsizex) x (width / bsizey) x 1
 */
template <typename TensorDataType>
__global__ void bp_training_error_signal_kernel(size_t height,
                                                size_t width,
                                                TensorDataType epsilon,
                                                unsigned long long statistics_count,
                                                const TensorDataType* __restrict__ input,
                                                size_t input_ldim,
                                                const TensorDataType* __restrict__ gradient_wrt_output,
                                                size_t gradient_wrt_output_ldim,
                                                TensorDataType* __restrict__ gradient_wrt_input,
                                                size_t gradient_wrt_input_ldim,
                                                const TensorDataType* __restrict__ mean,
                                                const TensorDataType* __restrict__ var,
                                                const TensorDataType* __restrict__ gradient_wrt_mean,
                                                const TensorDataType* __restrict__ gradient_wrt_var) {
  const size_t gidx = threadIdx.x + blockIdx.x * blockDim.x;
  const size_t gidy = threadIdx.y + blockIdx.y * blockDim.y;
  const size_t nthreadsx = blockDim.x * gridDim.x;
  const size_t nthreadsy = blockDim.y * gridDim.y;
  for (size_t row = gidx; row < height; row += nthreadsx) {
    const auto& _mean = mean[row];
    const auto& _var = var[row];
    const auto& dmean = gradient_wrt_mean[row];
    const auto& dvar = gradient_wrt_var[row];
    const auto inv_stdev = cuda::rsqrt(_var + epsilon);
    for (size_t col = gidy; col < width; col += nthreadsy) {
      const auto& x = input[row + col * input_ldim];
      const auto& dy = gradient_wrt_output[row + col * gradient_wrt_output_ldim];
      auto& dx = gradient_wrt_input[row + col * gradient_wrt_input_ldim];
      dx = (dy * inv_stdev
            + dmean / TensorDataType(statistics_count)
            + dvar * (x - _mean) * TensorDataType(2) / TensorDataType(statistics_count - 1));
    }
  }
}

/** @brief Backprop for training.
 *
 *  Assumes forward prop uses mini-batch statistics. In other words,
 *  statistics are dependent on input.
 */
template <typename TensorDataType>
void bp_training_impl(lbann_comm& comm,
                      TensorDataType epsilon,
                      const El::AbstractDistMatrix<TensorDataType>& input,
                      const El::AbstractDistMatrix<TensorDataType>& gradient_wrt_output,
                      El::AbstractDistMatrix<TensorDataType>& gradient_wrt_input,
                      const El::AbstractDistMatrix<TensorDataType>& statistics,
                      El::AbstractDistMatrix<TensorDataType>& gradient_wrt_statistics) {

  // Local matrices
  const auto& local_input = dynamic_cast<const El::Matrix<TensorDataType, El::Device::GPU>&>(input.LockedMatrix());
  const auto& local_gradient_wrt_output = dynamic_cast<const El::Matrix<TensorDataType, El::Device::GPU>&>(gradient_wrt_output.LockedMatrix());
  auto& local_gradient_wrt_input = dynamic_cast<El::Matrix<TensorDataType, El::Device::GPU>&>(gradient_wrt_input.Matrix());
  const auto& local_statistics = dynamic_cast<const El::Matrix<TensorDataType, El::Device::GPU>&>(statistics.LockedMatrix());
  const auto local_mean = El::LockedView(local_statistics, El::ALL, El::IR(0));
  const auto local_var = El::LockedView(local_statistics, El::ALL, El::IR(1));
  auto& local_gradient_wrt_statistics = dynamic_cast<El::Matrix<TensorDataType, El::Device::GPU>&>(gradient_wrt_statistics.Matrix());
  auto local_gradient_wrt_mean = El::View(local_gradient_wrt_statistics, El::ALL, El::IR(0));
  auto local_gradient_wrt_var = El::View(local_gradient_wrt_statistics, El::ALL, El::IR(1));

  // Dimensions
  const size_t local_height = local_gradient_wrt_input.Height();
  const size_t local_width = local_gradient_wrt_input.Width();

  // Count for statistics
  // Note: Output is constant if statistics count is <=1, so error
  // signal is zero.
  /// @todo Local statistics
  /// @todo Arbitrary group sizes
  const size_t statistics_count = input.Width();
  if (statistics_count <= 1) {
    El::Zero(local_gradient_wrt_input);
    return;
  }

  // Compute local gradient w.r.t. batch statistics
  El::Zero(gradient_wrt_statistics);
  if (local_height > 0) {
    constexpr size_t block_size = 256;
    dim3 block_dims, grid_dims;
    block_dims.x = block_size;
    grid_dims.x = (local_height + block_size - 1) / block_size;
    bp_training_stats_gradient_kernel<TensorDataType>
      <<<grid_dims, block_dims, 0, El::GPUManager::Stream()>>>(
        local_height,
        local_width,
        epsilon,
        local_input.LockedBuffer(),
        local_input.LDim(),
        local_gradient_wrt_output.LockedBuffer(),
        local_gradient_wrt_output.LDim(),
        local_mean.LockedBuffer(),
        local_var.LockedBuffer(),
        local_gradient_wrt_mean.Buffer(),
        local_gradient_wrt_var.Buffer());
  }

  // Accumulate gradient w.r.t. statistics across processes
  /// @todo Local statistics
  /// @todo Arbitrary group sizes
  comm.allreduce(gradient_wrt_statistics,
                 gradient_wrt_statistics.RedundantComm(),
                 El::mpi::SUM);

  // Compute gradient w.r.t. input
  if (!local_input.IsEmpty()) {
    const size_t local_height = local_input.Height();
    const size_t local_width = local_input.Width();
    constexpr size_t block_size_x = 256;
    constexpr size_t block_size_y = 1;
    dim3 block_dims, grid_dims;
    block_dims.x = block_size_x;
    block_dims.y = block_size_y;
    grid_dims.x = (local_height + block_size_x - 1) / block_size_x;
    grid_dims.y = (local_width + block_size_y - 1) / block_size_y;
    bp_training_error_signal_kernel
      <<<grid_dims, block_dims, 0, El::GPUManager::Stream()>>>(
        local_height,
        local_width,
        epsilon,
        statistics_count,
        local_input.LockedBuffer(),
        local_input.LDim(),
        local_gradient_wrt_output.LockedBuffer(),
        local_gradient_wrt_output.LDim(),
        local_gradient_wrt_input.Buffer(),
        local_gradient_wrt_input.LDim(),
        local_mean.LockedBuffer(),
        local_var.LockedBuffer(),
        local_gradient_wrt_mean.LockedBuffer(),
        local_gradient_wrt_var.LockedBuffer());
  }

}

/**
 *  dL/dx_i = dL/dy_i / sqrt(var+epsilon)
 *
 *  Block dimensions: bsizex x bsizey x 1
 *
 *  Grid dimensions: (height / bsizex) x (width / bsizey) x 1
 */
template <typename TensorDataType>
__global__ void bp_inference_kernel(size_t height,
                                    size_t width,
                                    TensorDataType epsilon,
                                    const TensorDataType* __restrict__ gradient_wrt_output,
                                    size_t gradient_wrt_output_ldim,
                                    TensorDataType* __restrict__ gradient_wrt_input,
                                    size_t gradient_wrt_input_ldim,
                                    const TensorDataType* __restrict__ running_var) {
  const size_t gidx = threadIdx.x + blockIdx.x * blockDim.x;
  const size_t gidy = threadIdx.y + blockIdx.y * blockDim.y;
  const size_t nthreadsx = blockDim.x * gridDim.x;
  const size_t nthreadsy = blockDim.y * gridDim.y;
  for (size_t row = gidx; row < height; row += nthreadsx) {
    const auto& var = running_var[row];
    const auto inv_stdev = cuda::rsqrt(var + epsilon);
    for (size_t col = gidy; col < width; col += nthreadsy) {
      const auto& dy = gradient_wrt_output[row + col * gradient_wrt_output_ldim];
      auto& dx = gradient_wrt_input[row + col * gradient_wrt_input_ldim];
      dx = dy * inv_stdev;
    }
  }
}

/** @brief Backprop for inference.
 *
 *  Assumes forward prop uses running statistics. In other words,
 *  statistics are independent of input.
 */
template <typename TensorDataType>
void bp_inference_impl(DataType epsilon,
                       const El::AbstractDistMatrix<TensorDataType>& gradient_wrt_output,
                       El::AbstractDistMatrix<TensorDataType>& gradient_wrt_input,
                       const El::AbstractDistMatrix<TensorDataType>& running_var) {

  // Local matrices
  const auto& local_gradient_wrt_output = dynamic_cast<const El::Matrix<TensorDataType, El::Device::GPU>&>(gradient_wrt_output.LockedMatrix());
  auto& local_gradient_wrt_input = dynamic_cast<El::Matrix<TensorDataType, El::Device::GPU>&>(gradient_wrt_input.Matrix());
  const auto& local_running_var = dynamic_cast<const El::Matrix<TensorDataType, El::Device::GPU>&>(running_var.LockedMatrix());

  // Compute gradient w.r.t. input
  if (!local_gradient_wrt_output.IsEmpty()) {
    const size_t local_height = local_gradient_wrt_output.Height();
    const size_t local_width = local_gradient_wrt_output.Width();
    constexpr size_t block_size_x = 256;
    constexpr size_t block_size_y = 1;
    dim3 block_dims, grid_dims;
    block_dims.x = block_size_x;
    block_dims.y = block_size_y;
    grid_dims.x = (local_height + block_size_x - 1) / block_size_x;
    grid_dims.y = (local_width + block_size_y - 1) / block_size_y;
    bp_inference_kernel<TensorDataType>
      <<<grid_dims, block_dims, 0, El::GPUManager::Stream()>>>(
        local_height,
        local_width,
        epsilon,
        local_gradient_wrt_output.LockedBuffer(),
        local_gradient_wrt_output.LDim(),
        local_gradient_wrt_input.Buffer(),
        local_gradient_wrt_input.LDim(),
        local_running_var.LockedBuffer());
  }

}

template <typename TensorDataType>
void bp_impl(lbann_comm& comm,
             TensorDataType epsilon,
             bool is_training,
             const El::AbstractDistMatrix<TensorDataType>& input,
             const El::AbstractDistMatrix<TensorDataType>& gradient_wrt_output,
             El::AbstractDistMatrix<TensorDataType>& gradient_wrt_input,
             const El::AbstractDistMatrix<TensorDataType>& batch_statistics,
             El::AbstractDistMatrix<TensorDataType>& gradient_wrt_batch_statistics,
             const El::AbstractDistMatrix<TensorDataType>& running_var) {

  // Batchnorm has different behavior for training and inference
  if (is_training) {
    bp_training_impl<TensorDataType>(comm,
                                     epsilon,
                                     input,
                                     gradient_wrt_output,
                                     gradient_wrt_input,
                                     batch_statistics,
                                     gradient_wrt_batch_statistics);
  }
  else {
    bp_inference_impl<TensorDataType>(epsilon,
                                      gradient_wrt_output,
                                      gradient_wrt_input,
                                      running_var);
  }

}

} // namespace

// Template instantiation
template <typename TensorDataType, data_layout T_layout, El::Device Dev>
void entrywise_batch_normalization_layer<TensorDataType, T_layout, Dev>::fp_compute() {
  const auto mode = this->get_model()->get_execution_context().get_execution_mode();
  fp_impl(*this->get_comm(),
          this->m_decay,
          this->m_epsilon,
          mode == execution_mode::training,
          this->get_prev_activations(),
          this->get_activations(),
          *this->m_batch_statistics,
          this->get_data_type_weights(0).get_values(),
          this->get_data_type_weights(1).get_values());
}

template <typename TensorDataType, data_layout T_layout, El::Device Dev>
void entrywise_batch_normalization_layer<TensorDataType, T_layout, Dev>::bp_compute() {
  const auto mode = this->get_model()->get_execution_context().get_execution_mode();
  bp_impl(*this->get_comm(),
          this->m_epsilon,
          mode == execution_mode::training,
          this->get_prev_activations(),
          this->get_prev_error_signals(),
          this->get_error_signals(),
          *this->m_batch_statistics,
          *this->m_batch_statistics_gradient,
          this->get_data_type_weights(1).get_values());
}

#define PROTO(T)                                      \
  template class entrywise_batch_normalization_layer< \
    T, data_layout::DATA_PARALLEL, El::Device::GPU>;  \
  template class entrywise_batch_normalization_layer< \
    T, data_layout::MODEL_PARALLEL, El::Device::GPU>

#define LBANN_INSTANTIATE_GPU_HALF
#include "lbann/macros/instantiate.hpp"

} // namespace lbann
