#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// Copyright (c) 2014-2019, Lawrence Livermore National Security, LLC.
// Produced at the Lawrence Livermore National Laboratory.
// Written by the LBANN Research Team (B. Van Essen, et al.) listed in
// the CONTRIBUTORS file. <lbann-dev@llnl.gov>
//
// LLNL-CODE-697807.
// All rights reserved.
//
// This file is part of LBANN: Livermore Big Artificial Neural Network
// Toolkit. For details, see http://software.llnl.gov/LBANN or
// https://github.com/LLNL/LBANN.
//
// Licensed under the Apache License, Version 2.0 (the "Licensee"); you
// may not use this file except in compliance with the License.  You may
// obtain a copy of the License at:
//
// http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or
// implied. See the License for the specific language governing
// permissions and limitations under the license.
////////////////////////////////////////////////////////////////////////////////

#define LBANN_BATCH_NORMALIZATION_LAYER_INSTANTIATE
#include "lbann/layers/regularizers/batch_normalization.hpp"
#include "lbann/utils/cuda.hpp"

namespace lbann {

namespace {

/** CUDA kernel to compute channel sums.
 *  Sums and squares of sums are used to compute mean and variance.
 */
template <El::Int block_size, typename TensorDataType>
__global__ void channel_sums_kernel(
  El::Int channel_height,
  El::Int width,
  const TensorDataType * __restrict__ data, El::Int data_ldim,
        TensorDataType * __restrict__ sums,
        TensorDataType * __restrict__ sqsums) {

  // Indices
  const El::Int tid = threadIdx.x;
  const El::Int gidx = threadIdx.x + blockIdx.x * blockDim.x;
  const El::Int bidy = blockIdx.y;

  // Initialize shared memory
  __shared__ TensorDataType shared_sums[block_size];
  __shared__ TensorDataType shared_sqsums[block_size];

  // Compute row sums in shared memory
  TensorDataType private_sum = 0;
  TensorDataType private_sqsum = 0;
  if (gidx < channel_height) {
    const auto& row = gidx + bidy * channel_height;
    for (El::Int col = 0; col < width; ++col) {
      const auto& x = data[row + col * data_ldim];
      private_sum += x;
      private_sqsum += x * x;
    }
  }
  shared_sums[tid] = private_sum;
  shared_sqsums[tid] = private_sqsum;

  // Compute channel sum with shared memory reduction
  /// @todo unroll loops
  for (El::Int stride = block_size / 2; stride > 0; stride /= 2) {
    __syncthreads();
    if(tid < stride) {
      shared_sums[tid] += shared_sums[tid + stride];
      shared_sqsums[tid] += shared_sqsums[tid + stride];
    }
  }

  // Output channel sum to global memory
  if (tid == 0) {
    cuda::atomic_add(&sums[bidy], shared_sums[0]);
    cuda::atomic_add(&sqsums[bidy], shared_sqsums[0]);
  }

}

/** CUDA kernel to compute statistics.
 *  On input, global_mean and global_var are assumed to contain sums
 *  and squares of sums, respectively.
 */
template <typename TensorDataType>
__global__ void compute_statistics_kernel(
  El::Int num_sums,
  El::Int num_per_sum,
  TensorDataType epsilon,
  TensorDataType decay,
  TensorDataType * __restrict__ global_mean,
  TensorDataType * __restrict__ global_var,
  TensorDataType * __restrict__ global_running_mean,
  TensorDataType * __restrict__ global_running_var) {

  const El::Int gid = threadIdx.x + blockIdx.x * blockDim.x;
  const El::Int num_threads = blockDim.x * gridDim.x;
  for (El::Int i = gid; i < num_sums; i += num_threads) {

    TensorDataType num_per_sum_dt = TensorDataType(num_per_sum);
    // Compute mean and variance
    const auto& mean = global_mean[i] / num_per_sum_dt;
    const auto& sqmean = global_var[i] / num_per_sum_dt;
    auto var = num_per_sum_dt * (sqmean - mean * mean) / TensorDataType(num_per_sum - 1);
    var = var > epsilon ? var : epsilon;
    global_mean[gid] = mean;
    global_var[gid] = var;

    // Compute running statistics
    auto& running_mean = global_running_mean[gid];
    auto& running_var = global_running_var[gid];
    running_mean = decay * running_mean + (TensorDataType(1.0) - decay) * mean;
    running_var = decay * running_var + (TensorDataType(1.0) - decay) * var;

  }

}

/** CUDA kernel to apply batch normalization. */
template <El::Int block_size, typename TensorDataType>
__global__ void batch_normalization_kernel(
  El::Int channel_height,
  El::Int width,
  const TensorDataType * __restrict__ global_input, El::Int input_ldim,
  const TensorDataType * __restrict__ global_mean,
  const TensorDataType * __restrict__ global_var,
  TensorDataType epsilon,
  const TensorDataType * __restrict__ global_scale,
  const TensorDataType * __restrict__ global_bias,
        TensorDataType * __restrict__ global_output, El::Int output_ldim) {

  // Indices
  const El::Int gidx = threadIdx.x + blockIdx.x * blockDim.x;
  const El::Int bidy = blockIdx.y;

  // Copy batch normalization parameters to private memory
  const auto& mean = global_mean[bidy];
  const auto& var = global_var[bidy];
  const auto& scale = global_scale[bidy];
  const auto& bias = global_bias[bidy];

  // Get reciprocal of standard deviation
  const auto& inv_stdev = cuda::rsqrt(var + epsilon);

  // Apply batch normalization
  if (gidx < channel_height) {
    const auto& row = gidx + bidy * channel_height;
    for (El::Int col = 0; col < width; ++col) {
      const auto& x = global_input[row + col * input_ldim];
      const auto& xhat = (x - mean) * inv_stdev;
      const auto& y = scale * xhat + bias;
      global_output[row + col * output_ldim] = y;
    }
  }

}

/** CUDA kernel to compute gradients w.r.t. batch norm parameters. */
template <El::Int block_size, typename TensorDataType>
__global__ void backprop1_kernel(
  El::Int channel_height,
  El::Int width,
  const TensorDataType * __restrict__ global_input,
  El::Int input_ldim,
  const TensorDataType * __restrict__ global_gradient_wrt_output,
  El::Int gradient_wrt_output_ldim,
  const TensorDataType * __restrict__ global_mean,
  const TensorDataType * __restrict__ global_var,
  TensorDataType epsilon,
  const TensorDataType * __restrict__ global_scale,
        TensorDataType * __restrict__ global_dscale,
        TensorDataType * __restrict__ global_dbias,
        TensorDataType * __restrict__ global_dmean,
        TensorDataType * __restrict__ global_dvar) {

  // Indices
  const El::Int tid = threadIdx.x;
  const El::Int gidx = threadIdx.x + blockIdx.x * blockDim.x;
  const El::Int bidy = blockIdx.y;

  // Initialize shared memory
  __shared__ TensorDataType shared_dscale[block_size];
  __shared__ TensorDataType shared_dbias[block_size];
  __shared__ TensorDataType shared_dmean[block_size];
  __shared__ TensorDataType shared_dvar[block_size];

  // Copy batch normalization parameters to private memory
  const auto& mean = global_mean[bidy];
  const auto& var = global_var[bidy];
  const auto& scale = global_scale[bidy];

  // Compute useful constants
  const TensorDataType zero = TensorDataType(0);
  const auto& inv_stdev = cuda::rsqrt(var + epsilon);
  const auto& dvar_factor = inv_stdev * inv_stdev * inv_stdev / TensorDataType(2);

  // Compute row-wise gradient contributions in shared memory
  auto dscale = zero;
  auto dbias = zero;
  auto dmean = zero;
  auto dvar = zero;
  if (gidx < channel_height) {
    const auto& row = gidx + bidy * channel_height;
    for(El::Int col = 0; col < width; ++col) {
      const auto& x = global_input[row + col * input_ldim];
      const auto& xhat = (x - mean) * inv_stdev;
      const auto& dy = global_gradient_wrt_output[row + col * gradient_wrt_output_ldim];
      dscale += dy * xhat;
      dbias += dy;
      const auto& dxhat = dy * scale;
      dmean += - dxhat * inv_stdev;
      dvar += - dxhat * (x - mean) * dvar_factor;
    }
  }
  shared_dscale[tid] = dscale;
  shared_dbias[tid] = dbias;
  shared_dmean[tid] = dmean;
  shared_dvar[tid] = dvar;

  // Compute gradients with shared memory reduction
  // @todo unroll loops
  for (El::Int stride = block_size / 2; stride > 0; stride /= 2) {
    __syncthreads();
    if (tid < stride) {
      shared_dscale[tid] += shared_dscale[tid + stride];
      shared_dbias[tid] += shared_dbias[tid + stride];
      shared_dmean[tid] += shared_dmean[tid + stride];
      shared_dvar[tid] += shared_dvar[tid + stride];
    }
  }

  // Output channel sum to global memory
  if (tid == 0) {
    cuda::atomic_add(&global_dscale[bidy], shared_dscale[0]);
    cuda::atomic_add(&global_dbias[bidy], shared_dbias[0]);
    cuda::atomic_add(&global_dmean[bidy], shared_dmean[0]);
    cuda::atomic_add(&global_dvar[bidy], shared_dvar[0]);
  }

}

/** CUDA kernel to compute gradients w.r.t. input. */
template <El::Int block_size, typename TensorDataType>
__global__ void backprop2_kernel(
  El::Int channel_height,
  El::Int local_width,
  El::Int num_per_sum,
  const TensorDataType * __restrict__ global_input,
  El::Int input_ldim,
  const TensorDataType * __restrict__ global_gradient_wrt_output,
  El::Int gradient_wrt_output_ldim,
  const TensorDataType * __restrict__ global_mean,
  const TensorDataType * __restrict__ global_var,
  TensorDataType epsilon,
  const TensorDataType * __restrict__ global_scale,
  const TensorDataType * __restrict__ global_dmean,
  const TensorDataType * __restrict__ global_dvar,
        TensorDataType * __restrict__ global_gradient_wrt_input,
  El::Int gradient_wrt_input_ldim) {

  // Indices
  const El::Int gidx = threadIdx.x + blockIdx.x * blockDim.x;
  const El::Int bidy = blockIdx.y;

  // Copy batch normalization parameters to private memory
  const auto& mean = global_mean[bidy];
  const auto& var = global_var[bidy];
  const auto& scale = global_scale[bidy];
  const auto& dmean = global_dmean[bidy];
  const auto& dvar = global_dvar[bidy];

  // Compute useful constants
  const auto& inv_stdev = cuda::rsqrt(var + epsilon);
  const auto& dmean_term = dmean / TensorDataType(num_per_sum);
  const auto& dvar_term = dvar * TensorDataType(2) / TensorDataType(num_per_sum - 1);

  // Apply batch normalization
  if (gidx < channel_height) {
    const auto& row = gidx + bidy * channel_height;
    for (El::Int col = 0; col < local_width; ++col) {
      const auto& x = global_input[row + col * input_ldim];
      const auto& dy = global_gradient_wrt_output[row + col * gradient_wrt_output_ldim];
      const auto& dxhat = dy * scale;
      auto& dx = global_gradient_wrt_input[row + col * gradient_wrt_input_ldim];
      dx = dxhat * inv_stdev + dmean_term + dvar_term * (x - mean);
    }
  }

}

} // namespace

template <typename TensorDataType, data_layout T_layout, El::Device Dev>
void batch_normalization_layer<TensorDataType, T_layout, Dev>::fp_compute() {

  const bool is_training = this->m_model->get_execution_context().get_execution_mode() == execution_mode::training;

  // CUDA objects
  CHECK_CUDA(hipSetDevice(El::GPUManager::Device()));
  auto&& stream = El::GPUManager::Stream();

  // Matrices
  const auto& input = this->get_prev_activations();
  const auto& local_input = input.LockedMatrix();
  auto& local_output = this->get_local_activations();

  // Matrix parameters
  const auto& width = input.Width();
  const auto& local_width = local_input.Width();
  const auto& output_dims = this->get_output_dims();
  const auto& num_channels = output_dims[0];
  const auto& channel_size = this->get_output_size() / num_channels;

  // Compute statistics
  if (is_training) {

    // Local matrices
    auto& local_mean = this->m_mean_v->Matrix();
    auto& local_var = this->m_var_v->Matrix();
    auto& local_running_mean = this->get_data_type_weights(2).get_values().Matrix();
    auto& local_running_var = this->get_data_type_weights(3).get_values().Matrix();

    // Compute sums and sums of squares
    El::Zero(local_mean);
    El::Zero(local_var);
    if (!local_input.IsEmpty()) {
      const El::Int block_size = 256;
      dim3 block_dims, grid_dims;
      block_dims.x = block_size;
      grid_dims.x = (channel_size + block_size - 1) / block_size;
      grid_dims.y = num_channels;
      channel_sums_kernel<block_size>
        <<<grid_dims, block_dims, 0, stream>>>(
          channel_size, local_width,
          local_input.LockedBuffer(), local_input.LDim(),
          local_mean.Buffer(), local_var.Buffer());
    }
    El::Int num_per_sum;
    if (this->m_statistics_group_size == 0) {
      // Global statistics aggregation; allreduce on fused buffer.
      this->m_comm->allreduce(*this->m_mean_and_var, this->m_mean_and_var->RedundantComm(),
                        El::mpi::SUM);
      num_per_sum = channel_size * width;
    } else if (this->m_statistics_group_size == 1) {
      // Local aggregation, no allreduce needed.
      num_per_sum = channel_size * local_width;
    } else {
      // Grouped batchnorm. Allreduce on fused buffer.
      this->m_comm->allreduce(*this->m_mean_and_var,
                        this->m_comm->get_packed_group_comm(this->m_statistics_group_size),
                        El::mpi::SUM);
      if (this->m_num_per_sum_cache.count(width) == 0) {
        num_per_sum = channel_size * local_width;
        num_per_sum = this->m_comm->allreduce(
          num_per_sum, this->m_comm->get_packed_group_comm(this->m_statistics_group_size));
        this->m_num_per_sum_cache[width] = num_per_sum;
      } else {
        num_per_sum = this->m_num_per_sum_cache[width];
      }
    }

    // Compute minibatch statistics
    if (num_per_sum <= 1) {
      El::Fill(local_var, TensorDataType(1.0));
    } else if (num_channels > 0) {
      const El::Int block_dim = 256;
      const El::Int grid_dim = (num_channels + block_dim - 1) / block_dim;
      compute_statistics_kernel<<<grid_dim, block_dim, 0, stream>>>(
          num_channels, num_per_sum, this->m_epsilon, this->m_decay,
          local_mean.Buffer(), local_var.Buffer(),
          local_running_mean.Buffer(), local_running_var.Buffer());
    }

  }

  // Apply batch normalization
  const auto& local_scale = this->get_data_type_weights(0).get_values().LockedMatrix();
  const auto& local_bias = this->get_data_type_weights(1).get_values().LockedMatrix();
  const auto& local_mean = (is_training ?
                            this->m_mean_v->LockedMatrix() :
                            this->get_data_type_weights(2).get_values().LockedMatrix());
  const auto& local_var = (is_training ?
                           this->m_var_v->LockedMatrix() :
                           this->get_data_type_weights(3).get_values().LockedMatrix());
  if (!local_input.IsEmpty()) {
    const El::Int block_size = 256;
    dim3 block_dims, grid_dims;
    block_dims.x = block_size;
    grid_dims.x = (channel_size + block_size - 1) / block_size;
    grid_dims.y = num_channels;
    batch_normalization_kernel<block_size>
      <<<grid_dims, block_dims, 0, stream>>>(
        channel_size, local_width,
        local_input.LockedBuffer(), local_input.LDim(),
        local_mean.LockedBuffer(), local_var.LockedBuffer(), this->m_epsilon,
        local_scale.LockedBuffer(), local_bias.LockedBuffer(),
        local_output.Buffer(), local_output.LDim());
  }

}

template <typename TensorDataType, data_layout T_layout, El::Device Dev>
void batch_normalization_layer<TensorDataType, T_layout, Dev>::bp_compute() {

  const bool is_training = this->m_model->get_execution_context().get_execution_mode() == execution_mode::training;

  // CUDA objects
  CHECK_CUDA(hipSetDevice(El::GPUManager::Device()));
  auto&& stream = El::GPUManager::Stream();

  // Matrices
  const auto& local_scale = this->get_data_type_weights(0).get_values().LockedMatrix();
  const auto& local_mean = (is_training ?
                            this->m_mean_v->LockedMatrix() :
                            this->get_data_type_weights(2).get_values().LockedMatrix());
  const auto& local_var = (is_training ?
                           this->m_var_v->LockedMatrix() :
                           this->get_data_type_weights(3).get_values().LockedMatrix());
  const auto& input = this->get_prev_activations();
  const auto& local_input = input.LockedMatrix();
  const auto& local_gradient_wrt_output = this->get_local_prev_error_signals();
  auto& local_gradient_wrt_input = this->get_local_error_signals();
  auto& local_mean_gradient = this->m_mean_gradient_v->Matrix();
  auto& local_var_gradient = this->m_var_gradient_v->Matrix();
  auto& local_scale_gradient = this->m_scale_gradient->Matrix();
  auto& local_bias_gradient = this->m_bias_gradient->Matrix();

  // Matrix parameters
  const auto& width = input.Width();
  const auto& local_width = local_input.Width();
  const auto& output_dims = this->get_output_dims();
  const auto& num_channels = output_dims[0];
  const auto& channel_size = this->get_output_size() / num_channels;

  // Compute local gradients
  // Compute gradients w.r.t. batch norm parameters
  El::Zero(local_scale_gradient);
  El::Zero(local_bias_gradient);
  El::Zero(local_mean_gradient);
  El::Zero(local_var_gradient);
  if (!local_input.IsEmpty()) {
    const El::Int block_size = 256;
    dim3 block_dims, grid_dims;
    block_dims.x = block_size;
    grid_dims.x = (channel_size + block_size - 1) / block_size;
    grid_dims.y = num_channels;
    backprop1_kernel<block_size>
      <<<grid_dims, block_dims, 0, stream>>>(
        channel_size, local_width,
        local_input.LockedBuffer(), local_input.LDim(),
        local_gradient_wrt_output.LockedBuffer(), local_gradient_wrt_output.LDim(),
        local_mean.LockedBuffer(), local_var.LockedBuffer(), this->m_epsilon,
        local_scale.LockedBuffer(),
        local_scale_gradient.Buffer(), local_bias_gradient.Buffer(),
        local_mean_gradient.Buffer(), local_var_gradient.Buffer());
  }

  // Accumulate gradients
  if (is_training) {
    if (this->m_statistics_group_size == 0) {
      // Global aggregation; allreduce on fused buffer.
      this->m_comm->allreduce(*this->m_mean_and_var_gradient,
                        this->m_mean_and_var_gradient->RedundantComm(),
                        El::mpi::SUM);
    } else if (this->m_statistics_group_size > 1) {
      // Grouped batchnorm; allreduce on fused buffer.
      this->m_comm->allreduce(*this->m_mean_and_var_gradient,
                        this->m_comm->get_packed_group_comm(this->m_statistics_group_size),
                        El::mpi::SUM);
    }
  } else {
    // Zero fused buffer.
    El::Zero(*this->m_mean_and_var_gradient);
  }
  auto* scale_optimizer = this->get_data_type_weights(0).get_optimizer();
  if (scale_optimizer != nullptr) {
    scale_optimizer->add_to_gradient(*this->m_scale_gradient, TensorDataType(1.0), true);
  }
  auto* bias_optimizer = this->get_data_type_weights(1).get_optimizer();
  if (bias_optimizer != nullptr) {
    bias_optimizer->add_to_gradient(*this->m_bias_gradient, TensorDataType(1.0), true);
  }

  // Compute error signal
  El::Int num_per_sum;
  if (this->m_statistics_group_size == 0) {
    // Global statistics aggregation.
    num_per_sum = channel_size * width;
  } else if (this->m_statistics_group_size == 1) {
    // Local aggregation.
    num_per_sum = channel_size * local_width;
  } else {
    // Grouped batchnorm.
    num_per_sum = this->m_num_per_sum_cache[width];  // This was computed in FP.
  }
  if (num_per_sum <= 1) {
    El::Zero(local_gradient_wrt_input);
  } else if (!local_input.IsEmpty()) {
    const El::Int block_size = 256;
    dim3 block_dims, grid_dims;
    block_dims.x = block_size;
    grid_dims.x = (channel_size + block_size - 1) / block_size;
    grid_dims.y = num_channels;
    backprop2_kernel<block_size>
      <<<grid_dims, block_dims, 0, stream>>>(
        channel_size, local_width, num_per_sum,
        local_input.LockedBuffer(), local_input.LDim(),
        local_gradient_wrt_output.LockedBuffer(), local_gradient_wrt_output.LDim(),
        local_mean.LockedBuffer(), local_var.LockedBuffer(), this->m_epsilon,
        local_scale.LockedBuffer(),
        local_mean_gradient.LockedBuffer(), local_var_gradient.LockedBuffer(),
        local_gradient_wrt_input.Buffer(), local_gradient_wrt_input.LDim());
  }

}

#define PROTO(T)                                      \
  template class batch_normalization_layer<T, data_layout::DATA_PARALLEL, El::Device::GPU>

#define LBANN_INSTANTIATE_GPU_HALF
#include "lbann/macros/instantiate.hpp"

} // namespace lbann
