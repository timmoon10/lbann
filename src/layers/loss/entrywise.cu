#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// Copyright (c) 2014-2019, Lawrence Livermore National Security, LLC.
// Produced at the Lawrence Livermore National Laboratory.
// Written by the LBANN Research Team (B. Van Essen, et al.) listed in
// the CONTRIBUTORS file. <lbann-dev@llnl.gov>
//
// LLNL-CODE-697807.
// All rights reserved.
//
// This file is part of LBANN: Livermore Big Artificial Neural Network
// Toolkit. For details, see http://software.llnl.gov/LBANN or
// https://github.com/LLNL/LBANN.
//
// Licensed under the Apache License, Version 2.0 (the "Licensee"); you
// may not use this file except in compliance with the License.  You may
// obtain a copy of the License at:
//
// http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or
// implied. See the License for the specific language governing
// permissions and limitations under the license.
////////////////////////////////////////////////////////////////////////////////

#define LBANN_ENTRYWISE_LAYER_INSTANTIATE
#include "lbann/layers/loss/entrywise.hpp"
#include "lbann/utils/cuda.hpp"

namespace lbann {

namespace {

/** CUDA kernel to apply an binary backprop operator. */
template <template <typename> class BinaryBackPropOperator,
          typename TensorDataType>
__global__
void binary_backprop_operator_kernel(El::Int height, El::Int width,
                                     const TensorDataType* __restrict__ x1,
                                     El::Int x1_ldim,
                                     const TensorDataType* __restrict__ x2,
                                     El::Int x2_ldim,
                                     const TensorDataType* __restrict__ dy,
                                     El::Int dy_ldim,
                                     TensorDataType* __restrict__ dx1,
                                     El::Int dx1_ldim,
                                     TensorDataType* __restrict__ dx2,
                                     El::Int dx2_ldim) {
  const El::Int gid = threadIdx.x + blockIdx.x * blockDim.x;
  const El::Int size = height * width;
  const El::Int num_threads = blockDim.x * gridDim.x;
  BinaryBackPropOperator<TensorDataType> op;
  for (El::Int pos = gid; pos < size; pos += num_threads) {
    const auto& row = pos % height;
    const auto& col = pos / height;
    op(x1[row + col * x1_ldim],
       x2[row + col * x2_ldim],
       dy[row + col * dy_ldim],
       dx1[row + col * dx1_ldim],
       dx2[row + col * dx2_ldim]);
  }
}

/** Apply a binary backprop operator to CPU data.
 *  The input and output data must be on CPU and must have the same
 *  dimensions. Given a binary function \f$ y = f(x_1,x_2) \f$, the
 *  corresponding BinaryBackPropOperator is a 5-ary function with the
 *  arguments \f$ x_1 \f$, \f$ x_2 \f$, \f$ dL/dy \f$, \f$ dL/dx_1\f$,
 *  \f$ dL/dx_2 \f$. The last two arguments should be overwritten when
 *  the BinaryBackPropOperator is called.
 */
template <template <typename> class Op, typename TensorDataType>
void apply_binary_backprop_operator(
  const El::AbstractMatrix<TensorDataType>& x1,
  const El::AbstractMatrix<TensorDataType>& x2,
  const El::AbstractMatrix<TensorDataType>& dy,
  El::AbstractMatrix<TensorDataType>& dx1,
  El::AbstractMatrix<TensorDataType>& dx2) {

  // Get CUDA grid dimensions
  // Note: Maximum CUDA grid dimension is 2^32-1
  // (https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#features-and-technical-specifications).
  const El::Int height = x1.Height();
  const El::Int width = x1.Width();
  const El::Int block_dim = 256;
  El::Int grid_dim = (height * width + block_dim - 1) / block_dim;
  if (sizeof(El::Int) > sizeof(unsigned int)
      && grid_dim > std::numeric_limits<uint32_t>::max()) {
    grid_dim = std::numeric_limits<uint32_t>::max();
  }

  // Launch CUDA kernel
  if (grid_dim > 0) {
    CHECK_CUDA(hipSetDevice(El::GPUManager::Device()));
    binary_backprop_operator_kernel<Op>
      <<<grid_dim, block_dim, 0, El::GPUManager::Stream()>>>(
        height, width,
        x1.LockedBuffer(), x1.LDim(),
        x2.LockedBuffer(), x2.LDim(),
        dy.LockedBuffer(), dy.LDim(),
        dx1.Buffer(), dx1.LDim(),
        dx2.Buffer(), dx2.LDim());
  }

}

// =========================================================
// Operator objects for entry-wise binary layers
// =========================================================
// Note: Binary operator corresponds to forward prop step
// (\f$ y = f(x_1,x_2) \f$) and 5-ary operator corresponds
// to back prop step
// (\f$ \frac{dL}{dx_i} = \frac{dL}{dy} \frac{df}{dx_i}(x_1,x_2) \f$).

/** Binary cross entropy operator. */
template <typename TensorDataType>
struct binary_cross_entropy_op {
  inline __device__ TensorDataType operator()(const TensorDataType& x1,
                                        const TensorDataType& x2) const {
    const TensorDataType zero = 0.;
    const TensorDataType one = 1.;
    TensorDataType y = zero;
    if (x2 > zero) { y += -x2 * cuda::log(x1); }
    if (x2 < one)  { y += -(one-x2) * cuda::log(one-x1); }
    return y;
  }
  inline __device__ void operator()(const TensorDataType& x1,
                                    const TensorDataType& x2,
                                    const TensorDataType& dy,
                                    TensorDataType& dx1,
                                    TensorDataType& dx2) const {
    const TensorDataType zero = 0.;
    const TensorDataType one = 1.;
    dx1 = zero;
    dx2 = zero;
    if (dy == zero) { return; }
    if (x2 > zero) {
      dx1 += -x2 / x1 * dy;
      dx2 += -cuda::log(x1) * dy;
    }
    if (x2 < one)  {
      dx1 += (one-x2) / (one-x1) * dy;
      dx2 += cuda::log(one-x1) * dy;
    }
  }
};

/** Sigmoid binary cross entropy operator.
 *  Equivalent to applying a sigmoid function to the first operand and
 *  then computing the binary cross entropy. Numerically stable
 *  implementation is taken from
 *  https://www.tensorflow.org/api_docs/python/tf/nn/sigmoid_cross_entropy_with_logits.
 */
template <typename TensorDataType>
struct sigmoid_binary_cross_entropy_op {
  inline __device__ TensorDataType operator()(const TensorDataType& x1,
                                        const TensorDataType& x2) const {
    const TensorDataType zero = 0.;
    const TensorDataType one = 1.;
    const auto& z = cuda::max(zero, cuda::min(x2, one));
    if (x1 > zero) {
      return (one - z) * x1 + cuda::log1p(cuda::exp(-x1));
    } else {
      return - x1 * z + cuda::log1p(cuda::exp(x1));
    }
  }
  inline __device__ void operator()(const TensorDataType& x1,
                                    const TensorDataType& x2,
                                    const TensorDataType& dy,
                                    TensorDataType& dx1,
                                    TensorDataType& dx2) const {
    const TensorDataType zero = 0.;
    const TensorDataType one = 1.;
    const auto& z = cuda::max(zero, cuda::min(x2, one));
    if (x1 > zero) {
      dx1 = -z + one / (one + cuda::exp(-x1));
    } else {
      dx1 = one - z - one / (one + cuda::exp(x1));
    }
    dx1 *= dy;
    dx2 = (x2 == z) ? -x1 * dy : zero;
  }
};

/** Boolean accuracy operator. */
template <typename TensorDataType>
struct boolean_accuracy_op {
  inline __device__ TensorDataType operator()(const TensorDataType& x1,
                                        const TensorDataType& x2) const {
    const auto& b1 = x1 >= TensorDataType(0.5);
    const auto& b2 = x2 >= TensorDataType(0.5);
    return b1 == b2 ? TensorDataType(1.0) : TensorDataType(0.0);
  }
  inline __device__ void operator()(const TensorDataType& x1,
                                    const TensorDataType& x2,
                                    const TensorDataType& dy,
                                    TensorDataType& dx1,
                                    TensorDataType& dx2) const {
    dx1 = TensorDataType(0.0);
    dx2 = TensorDataType(0.0);
  }
};

/** Boolean false negative operator. */
template <typename TensorDataType>
struct boolean_false_negative_op {
  inline __device__ TensorDataType operator()(const TensorDataType& x1,
                                        const TensorDataType& x2) const {
    const auto& b1 = x1 >= TensorDataType(0.5);
    const auto& b2 = x2 >= TensorDataType(0.5);
    return (!b1 && b2) ? TensorDataType(1.0) : TensorDataType(0.0);
  }
  inline __device__ void operator()(const TensorDataType& x1,
                                    const TensorDataType& x2,
                                    const TensorDataType& dy,
                                    TensorDataType& dx1,
                                    TensorDataType& dx2) const {
    dx1 = TensorDataType(0.0);
    dx2 = TensorDataType(0.0);
  }
};

/** Boolean false positive operator. */
template <typename TensorDataType>
struct boolean_false_positive_op {
  inline __device__ TensorDataType operator()(const TensorDataType& x1,
                                        const TensorDataType& x2) const {
    const auto& b1 = x1 >= TensorDataType(0.5);
    const auto& b2 = x2 >= TensorDataType(0.5);
    return (b1 && !b2) ? TensorDataType(1.0) : TensorDataType(0.0);
  }
  inline __device__ void operator()(const TensorDataType& x1,
                                    const TensorDataType& x2,
                                    const TensorDataType& dy,
                                    TensorDataType& dx1,
                                    TensorDataType& dx2) const {
    dx1 = TensorDataType(0.0);
    dx2 = TensorDataType(0.0);
  }
};

} // namespace

// Template instantiation
#define DEFINE_COMPUTE_OPS(layer, op)                                   \
  template <typename TensorDataType, data_layout Layout, El::Device Device> \
  void layer<TensorDataType, Layout, Device>::fp_compute() {            \
    cuda::apply_entrywise_binary_operator<op>(                          \
      this->get_prev_activations(0),                                    \
      this->get_prev_activations(1),                                    \
      this->get_activations());                                         \
  }                                                                     \
  template <typename TensorDataType, data_layout Layout, El::Device Device> \
  void layer<TensorDataType, Layout, Device>::bp_compute() {            \
    apply_binary_backprop_operator<op>(                                 \
      this->get_local_prev_activations(0),                              \
      this->get_local_prev_activations(1),                              \
      this->get_local_prev_error_signals(),                             \
      this->get_local_error_signals(0),                                 \
      this->get_local_error_signals(1));                                \
  }                                                                     \

DEFINE_COMPUTE_OPS(binary_cross_entropy_layer, binary_cross_entropy_op)
DEFINE_COMPUTE_OPS(sigmoid_binary_cross_entropy_layer, sigmoid_binary_cross_entropy_op)
DEFINE_COMPUTE_OPS(boolean_accuracy_layer, boolean_accuracy_op)
DEFINE_COMPUTE_OPS(boolean_false_negative_layer, boolean_false_negative_op)
DEFINE_COMPUTE_OPS(boolean_false_positive_layer, boolean_false_positive_op)

#define PROTO(T) \
  BINARY_ETI_INST_MACRO_DEV_DT(binary_cross_entropy_layer, T, El::Device::GPU); \
  BINARY_ETI_INST_MACRO_DEV_DT(sigmoid_binary_cross_entropy_layer, T, El::Device::GPU); \
  BINARY_ETI_INST_MACRO_DEV_DT(boolean_accuracy_layer, T, El::Device::GPU); \
  BINARY_ETI_INST_MACRO_DEV_DT(boolean_false_negative_layer, T, El::Device::GPU); \
  BINARY_ETI_INST_MACRO_DEV_DT(boolean_false_positive_layer, T, El::Device::GPU)

#define LBANN_INSTANTIATE_GPU_HALF
#include "lbann/macros/instantiate.hpp"

} // namespace lbann
